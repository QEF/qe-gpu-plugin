#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2001-2013 Quantum ESPRESSO Foundation
 *
 * This file is distributed under the terms of the
 * GNU General Public License. See the file `License'
 * in the root directory of the present distribution,
 * or http://www.gnu.org/copyleft/gpl.txt .
 *
 */

#include <stdlib.h>
#include <stdio.h>

#include "cuda_env.h"

typedef double fftw_complex[2];

__global__ void kernel_vec_prod_k( double *a, const  double * __restrict b, int dimx )
{	   
	register int ix = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	register double sup;
	register int ii = ix / 2;

	if ( ix < ( dimx * 2 ) ) {
		sup = a[ix] * b[ii];
		a[ix] = sup;
	}
}

__global__ void build_psic_index(const  int * __restrict nls, const  int * __restrict igk, int * psic_index_nls, const int n ){

	register int ix = blockDim.x * blockIdx.x + threadIdx.x;

	if ( ix < n ) {

		// TODO: Fetch in shared memory igk[ ix ]
		// TODO: In-place index calculation

		psic_index_nls[ix] = ( nls[ igk[ ix ] - 1 ] - 1 ) * 2;

		// TODO: Copy from shared to global memory

	}
}

__global__ void kernel_init_psic_k( const  int * __restrict psic_index_nls,  const  double * __restrict psi, double *psic, const int n, const int lda, const int ibnd )
{	   
	register int ix = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	register int psi_index = ( ix + ( ibnd * lda ) ) * 2;

	if ( ix < n ) {
		//psic_index_nls = ( nls[ igk[ ix ] - 1 ] - 1 ) * 2;

		psic[ psic_index_nls[ix] ] = psi[ psi_index ];
		psic[ psic_index_nls[ix] + 1 ] = psi[ psi_index + 1 ];
	}
}

__global__ void kernel_save_hpsi_k( const  int * __restrict psic_index_nls, double * hpsi, const  double * __restrict psic, const int n, const int ibnd )
{	   
	register int ix = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	register int psi_index = (ix + (ibnd * n)) * 2;

	if ( ix < (n) ) {
		// psic_index_nls = (nls[ igk[ ix ] - 1 ] - 1) * 2;
		hpsi[ psi_index ] = hpsi[ psi_index ] + psic[ psic_index_nls[ix] ];
		hpsi[ psi_index + 1 ] = hpsi[ psi_index + 1 ] + psic[ psic_index_nls[ix] + 1 ];
	}
}


extern "C" int vloc_psi_cuda_k_( int * ptr_lda, int * ptr_nrxxs, int * ptr_nr1s, int * ptr_nr2s, int * ptr_nr3s, int * ptr_n, int * ptr_m, hipfftDoubleComplex * psi, double * v, fftw_complex * hpsi, int * igk, int * nls, int * ptr_ngms)
{
	hipfftHandle p_global;
	fftw_complex * psic = NULL;

	void * psic_D, * psi_D; // hipfftDoubleComplex *
	void * v_D; // double *
	void * igk_D, * nls_D, * psic_index_nls_D; // int*

	int j,  blocksPerGrid, ibnd;
	double tscale;

	int n = (* ptr_n);
	int m = (* ptr_m);
	int nr1s = (* ptr_nr1s);
	int nr2s = (* ptr_nr2s);
	int nr3s = (* ptr_nr3s);
	int nrxxs = (* ptr_nrxxs);
	int ngms = (* ptr_ngms);
	int lda = (* ptr_lda);
#if defined(__CUDA_NOALLOC)
    int ierr;
#endif
    int size_psic = nr1s * nr2s * nr3s;

#if defined(__CUDA_DEBUG)
	printf("[VLOC_PSI_K_OPT] Enter (n=%d, m=%d, ngms=%d)\n", n, m, ngms); fflush(stdout);
#endif

	blocksPerGrid = ( n + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
	if ( blocksPerGrid > __CUDA_MAXNUMBLOCKS__) {
		fprintf( stderr, "\n[VLOC_PSI_K] kernel_init_psic_k cannot run, blocks requested ( %d ) > blocks allowed!!!", blocksPerGrid );
		return 1;
	}

	blocksPerGrid = ( (nrxxs * 2) + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
	if ( blocksPerGrid > __CUDA_MAXNUMBLOCKS__) {
		fprintf( stderr, "\n[VLOC_PSI_K] kernel_vec_prod cannot run, blocks requested ( %d ) > blocks allowed!!!", blocksPerGrid );
		return 1;
	}

	hipSetDevice(qe_gpu_bonded[0]);

#if defined(__CUDA_NOALLOC)
	/* Do real allocation */
	ierr = hipMalloc ( (void**) &(qe_dev_scratch[0]), (size_t) qe_gpu_mem_unused[0] );
    qecudaGenericErr((hipError_t) ierr, "VLOC_PSI_K_OPT", "error memory allocation (qe_dev_scratch)");

#if defined(__CUDA_KERNEL_MEMSET)
	qecudaSafeCall( hipMemset( qe_dev_scratch[0], 0, (size_t) qe_gpu_mem_unused[0] ) );
#endif

#endif

	size_t shift = 0;
	psic_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( size_psic )*sizeof( hipfftDoubleComplex );
	psi_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( lda * m )*sizeof( hipfftDoubleComplex );
	v_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( nrxxs )*sizeof( double );
	nls_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (ngms % 2 == 0)? ngms : ngms + 1 )*sizeof(int);
	igk_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (n % 2 == 0)? n : n + 1 )*sizeof(int);
	psic_index_nls_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (n % 2 == 0)? n : n + 1 )*sizeof(int);
	// now	shift contains the amount of byte required on the GPU to compute

	if ( shift > qe_gpu_mem_unused[0] ) {
		fprintf( stderr, "\n[VLOC_PSI_K] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", shift, qe_gpu_mem_unused[0] );
#if defined(__CUDA_NOALLOC)
		/* Deallocating... */
		ierr = hipFree ( qe_dev_scratch[0] );
	    qecudaGenericErr((hipError_t) ierr, "VLOC_PSI_K_OPT", "error memory release (qe_dev_scratch)");

#endif
		return 1;
	}

#if defined(__CUDA_KERNEL_MEMSET)
    qecudaSafeCall( hipMemset( psic_index_nls_D , 0, sizeof( int ) * n ) );
#endif
	qecudaSafeCall( hipMemcpy( psi_D, psi,  sizeof( hipfftDoubleComplex ) * lda * m, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( v_D, v,  sizeof( double ) * nrxxs, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( nls_D, nls,  sizeof( int ) * ngms, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( igk_D, igk,  sizeof( int ) * n, hipMemcpyHostToDevice ) );

	blocksPerGrid = ( n + __CUDA_TxB_VLOCPSI_BUILD_PSIC__ - 1) / __CUDA_TxB_VLOCPSI_BUILD_PSIC__ ;
    dim3 dimGrid(blocksPerGrid);
    dim3 dimBlock(__CUDA_TxB_VLOCPSI_BUILD_PSIC__);
    build_psic_index<<<dimGrid,dimBlock >>>( (int *) nls_D, (int *) igk_D, (int *) psic_index_nls_D, n );
	qecudaGetLastError("kernel launch failure");

#if defined(__CUDA_DEBUG)
	printf("[VLOC_PSI_K_OPT] psic_index_nls_D computed\n"); fflush(stdout);
#endif

	qecheck_cufft_call( hipfftPlan3d( &p_global, nr3s, nr2s,  nr1s, HIPFFT_Z2Z ) );
    qecheck_cufft_call( hipfftSetStream(p_global,qecudaStreams[ 0 ]) );

	qecudaSafeCall( hipHostAlloc ( (void**) &psic, size_psic * sizeof( fftw_complex ), hipHostMallocPortable ) );

	for ( ibnd =  0; ibnd < m; ibnd = ibnd + 1) {

		hipDeviceSynchronize();

		qecudaSafeCall( hipMemset( psic_D, 0, size_psic * sizeof( hipfftDoubleComplex ) ) );

		blocksPerGrid = ( n + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
		kernel_init_psic_k<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (int *) psic_index_nls_D, (double *) psi_D, (double *) psic_D, n, lda, ibnd );
		qecudaGetLastError("kernel launch failure");

		qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex *) psic_D, (hipfftDoubleComplex *) psic_D, HIPFFT_BACKWARD ) );

		blocksPerGrid = ( (nrxxs * 2) + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
		kernel_vec_prod_k<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (double *) psic_D, (double *) v_D , nrxxs );
		qecudaGetLastError("kernel launch failure");

		// VERIFY OVERLAP

		// schedule(static,chunk=64)
		if (ibnd > 0) {
#pragma omp for private(j)
			for ( j = 0; j <  n; j++ ) {
				hpsi[ j + ( ( ibnd  - 1 ) * lda ) ][0] += psic[ nls [ igk[ j ] - 1  ] - 1 ][0];
				hpsi[ j + ( ( ibnd  - 1 ) * lda ) ][1] += psic[ nls [ igk[ j ] - 1  ] - 1 ][1];
			}
		}
		
		qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex *) psic_D, (hipfftDoubleComplex *)psic_D, HIPFFT_FORWARD ) );

		tscale = 1.0 / (double) ( size_psic );

		hipblasZdscal(qecudaHandles[ 0 ] , size_psic, &tscale, (hipDoubleComplex *) psic_D, 1);

		qecudaSafeCall( hipMemcpy( psic, psic_D, sizeof( hipfftDoubleComplex ) * size_psic, hipMemcpyDeviceToHost ) );

//	    for( j = 0; j <  n; j++ ) {
//	      hpsi[ j + ( ibnd * lda ) ][0] += psic[ nls [ igk[ j ] - 1  ] - 1 ][0];
//	      hpsi[ j + ( ibnd * lda ) ][1] += psic[ nls [ igk[ j ] - 1  ] - 1 ][1];
//	    }

	}

#pragma omp for private(j)
	for ( j = 0; j <  n; j++ ) {
		hpsi[ j + ( ( m - 1 ) * lda ) ][0] += psic[ nls [ igk[ j ] - 1  ] - 1 ][0];
		hpsi[ j + ( ( m - 1 ) * lda ) ][1] += psic[ nls [ igk[ j ] - 1  ] - 1 ][1];
	}

	qecheck_cufft_call( hipfftDestroy(p_global) );

#if defined(__CUDA_NOALLOC)
	/* Deallocating... */
	ierr = hipFree ( qe_dev_scratch[0] );
    qecudaGenericErr((hipError_t) ierr, "VLOC_PSI_K_OPT", "error memory release (qe_dev_scratch)");

#else

#if defined(__CUDA_KERNEL_MEMSET)
	qecudaSafeCall( hipMemset( qe_dev_scratch[0], 0, (size_t) qe_gpu_mem_unused[0] ) );
#endif

#endif

#if defined(__CUDA_DEBUG)
	printf("[VLOC_PSI_K_OPT] Exit\n"); fflush(stdout);
#endif

	return 0;
}

#if defined(__CUDA_MULTIPLAN_FFT)
extern "C" void vloc_psi_multiplan_cuda_k_(  int * ptr_lda, int * ptr_nrxxs, int * ptr_nr1s, int * ptr_nr2s, int * ptr_nr3s, int * ptr_n, int * ptr_m, hipfftDoubleComplex * psi, double * v, fftw_complex * hpsi, int * igk, int * nls, int * ptr_ngms)
{
	void * psic_D, * psi_D; // hipfftDoubleComplex *
	fftw_complex * psic = NULL;
	void * v_D; // double *
	void * igk_D, * nls_D; // int *

	int blocksPerGrid;
	double tscale;

	int n = (* ptr_n);
	int m = (* ptr_m);
	int nr1s = (* ptr_nr1s);
	int nr2s = (* ptr_nr2s);
	int nr3s = (* ptr_nr3s);
	int nrxxs = (* ptr_nrxxs);
	int ngms = (* ptr_ngms);
	int lda = (* ptr_lda);
    int ierr;
	int dim_multiplepsic, n_singlepsic, n_multiplepsic, size_multiplepsic, i, j, k;
	int array[3];

	int size_psic = nr1s * nr2s * nr3s;

	int last;

	size_t buffer_size = 0L;

	hipSetDevice(qe_gpu_bonded[0]);

	dim_multiplepsic = __NUM_FFT_MULTIPLAN__;

	n_multiplepsic = m/dim_multiplepsic;
	n_singlepsic = m%dim_multiplepsic;

	size_multiplepsic = size_psic * dim_multiplepsic;

	buffer_size = size_multiplepsic * sizeof( hipfftDoubleComplex ) + sizeof( hipfftDoubleComplex ) * n * m + sizeof( int ) * ngms + sizeof( int ) * n + sizeof( double ) * nrxxs;

	if ( buffer_size > qe_gpu_mem_unused[0] ) {
		fprintf( stderr, "\n[VLOC_PSI_K] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", buffer_size, qe_gpu_mem_unused[0] );
		exit(EXIT_FAILURE);
	}

	size_t shift = 0;
	psic_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( size_psic )*sizeof( hipfftDoubleComplex );
	psi_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( lda * m )*sizeof( hipfftDoubleComplex );
	v_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( nrxxs )*sizeof( double );
	nls_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (ngms % 2 == 0)? ngms : ngms + 1 )*sizeof(int);
	igk_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (n % 2 == 0)? n : n + 1 )*sizeof(int);

	// now	shift contains the amount of byte required on the GPU to compute

	if ( shift > qe_gpu_mem_unused[0] ) {
		fprintf( stderr, "\n[VLOC_PSI_K] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", shift, qe_gpu_mem_unused[0] );
		exit(EXIT_FAILURE);
	}

	qecudaSafeCall( hipMemcpy( nls_D, nls,  sizeof( int ) * ngms, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( igk_D, igk,  sizeof( int ) * n, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( psi_D, psi,  sizeof( hipfftDoubleComplex ) * n * m, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( v_D, v,  sizeof( double ) * nrxxs, hipMemcpyHostToDevice ) );

	array[0] = nr3s;
	array[1] = nr2s;
	array[2] = nr1s;

	hipfftHandle p_global;

	if ( n_multiplepsic > 0 ) {

		qecheck_cufft_call( hipfftPlanMany( &p_global, 3, array, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, dim_multiplepsic ) );
        qecheck_cufft_call( hipfftSetStream(p_global,qecudaStreams[ 0 ]) );

		qecudaSafeCall( hipHostAlloc ( (void**) &psic, size_multiplepsic * sizeof( fftw_complex ), hipHostMallocPortable ) );

		for(j = 0; j< (m-n_singlepsic); j = j + dim_multiplepsic ) {

			qecudaSafeCall( hipMemset( psic_D, 0, size_psic * dim_multiplepsic * sizeof( hipfftDoubleComplex ) ) );

			blocksPerGrid = ( ( n * 2) + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
			for( i = 0; i < dim_multiplepsic; i++ ) {
				kernel_init_psic_k<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (int*) nls_D, (int*) igk_D, (double*) psi_D, (double*) psic_D, n, lda, (j+i));
				qecudaGetLastError("kernel launch failure");
			}

			qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D, HIPFFT_BACKWARD ) );

			blocksPerGrid = ( (nrxxs * 2) + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
			for( i = 0; i < dim_multiplepsic; i++ ) {
				kernel_vec_prod_k<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (double*) psic_D, (double*) v_D , nrxxs );
				qecudaGetLastError("kernel launch failure");
			}

			for( i = 0; i < dim_multiplepsic; i++ ) {
				if ( (i+j) > 0 ) {
					for ( k = 0; k <  n ; k++ ) {
						hpsi[ k + ( ( (i+j)  - 1 ) * lda ) ][0] += psic[ nls [ igk[ k ] - 1  ] - 1 ][0];
						hpsi[ k + ( ( (i+j)  - 1 ) * lda ) ][1] += psic[ nls [ igk[ k ] - 1  ] - 1 ][1];
					}
				}
			}

			qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D, HIPFFT_FORWARD ) );

			tscale = 1.0 / (double) ( size_psic );
			hipblasZdscal(qecudaHandles[ 0 ] , size_psic*dim_multiplepsic, &tscale, (hipDoubleComplex *) psic_D, 1);

			qecudaSafeCall( hipMemcpy( psic, (hipfftDoubleComplex*) psic_D, sizeof( hipfftDoubleComplex ) * size_psic * dim_multiplepsic, hipMemcpyDeviceToHost ) );
		}

		last = dim_multiplepsic*n_multiplepsic;
		for ( k = 0; k <  n; k++ ) {
			hpsi[ k + ( ( last - 1 ) * lda ) ][0] += psic[ nls [ igk[ k ] - 1  ] - 1 ][0];
			hpsi[ k + ( ( last - 1 ) * lda ) ][1] += psic[ nls [ igk[ k ] - 1  ] - 1 ][1];
		}

		qecheck_cufft_call( hipfftDestroy(p_global) );

		qecudaSafeCall( hipMemset( psic_D, 0, size_psic * dim_multiplepsic * sizeof( hipfftDoubleComplex ) ) );

		qecudaSafeCall( hipHostFree(psic));

	}

	if (n_singlepsic > 0 ) {

		printf("n_singlepsic\n");fflush(stdout);

		qecheck_cufft_call( hipfftPlanMany( &p_global, 3, array, NULL, 1, 0, NULL, 1, 0,HIPFFT_Z2Z, n_singlepsic ) );
        qecheck_cufft_call( hipfftSetStream(p_global,qecudaStreams[ 0 ]) );

		qecudaSafeCall( hipHostAlloc ( (void**) &psic, n_singlepsic * size_psic * sizeof( hipfftDoubleComplex ), hipHostMallocPortable ) );

		qecudaSafeCall( hipMemset( psic_D, 0, n_singlepsic * size_psic * sizeof( hipfftDoubleComplex ) ) );

		blocksPerGrid = ( ( n * 2) + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
		for( i = 0; i < n_singlepsic; i++ ) {
			shift = 2*i*size_psic*sizeof(double);
			kernel_init_psic_k<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (int*) nls_D, (int*) igk_D, (double*) psi_D, (double*) ( (char*) psic_D + shift ), n, lda, ((dim_multiplepsic*n_multiplepsic) +i) );
			qecudaGetLastError("kernel launch failure");
		}

		qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D, HIPFFT_BACKWARD ) );

		blocksPerGrid = ( (nrxxs * 2) + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
		for( i = 0; i < n_singlepsic; i++ ) {
			shift = 2*i*size_psic*sizeof(double);
			kernel_vec_prod_k<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (double *) ( (char*) psic_D + shift ), (double *) v_D , nrxxs );
			qecudaGetLastError("kernel launch failure");
		}

		qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D, HIPFFT_FORWARD ) );

		tscale = 1.0 / (double) ( size_psic );
		hipblasZdscal(qecudaHandles[ 0 ] , n_singlepsic * size_psic, &tscale, (hipDoubleComplex *) psic_D, 1);

		qecudaSafeCall( hipMemcpy( psic, psic_D, sizeof( hipfftDoubleComplex ) * n_singlepsic * size_psic, hipMemcpyDeviceToHost ) );

		for( i = 0; i < n_singlepsic; i++ ) {
			for( k = 0; k <  n; k++ ) {
				hpsi[ k + ( ((dim_multiplepsic*n_multiplepsic) + i) * lda ) ][0] += psic[ nls [ igk[ k ] - 1  ] - 1 ][0];
				hpsi[ k + ( ((dim_multiplepsic*n_multiplepsic) + i) * lda ) ][1] += psic[ nls [ igk[ k ] - 1  ] - 1 ][1];
			}
		}

		qecheck_cufft_call( hipfftDestroy(p_global) );
		qecudaSafeCall( hipHostFree(psic));
	}

}
#endif
