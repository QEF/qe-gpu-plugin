#include "hip/hip_runtime.h"
/*****************************************************************************\
 * Copyright (C) 2011-2014 Quantum ESPRESSO Foundation
 *
 * This file is distributed under the terms of the
 * GNU General Public License. See the file `License'
 * in the root directory of the present distribution,
 * or http://www.gnu.org/copyleft/gpl.txt .
 *
 * Filippo Spiga (filippo.spiga@quantum-espresso.org)
\*****************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include "cuda_env.h"

extern "C" void start_clock_(char * label, unsigned int length_arg );
extern "C" void stop_clock_(char * label, unsigned int length_arg );


__global__ void kernel_vec_prod( double *a, const  double * __restrict b, int dimx )
{	   
	register int ix = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	register double sup;
	register int ii = ix / 2;

	if( ix < ( dimx * 2 ) ){
		sup = a[ix] * b[ii];
		a[ix] = sup;
	}
}


__global__ void build_psic_index(const  int * __restrict nls, const  int * __restrict nlsm, const  int * __restrict igk, int * psic_index_nls, int * psic_index_nlsm, const int n ){

	register int ix = blockDim.x * blockIdx.x + threadIdx.x;

	if ( ix < n ) {

		// TODO: Fetch in shared memory igk[ ix ]
		// TODO: In-place index calculation

		psic_index_nls[ix] = ( nls[ igk[ ix ] - 1 ] - 1 ) * 2;
		psic_index_nlsm[ix] = ( nlsm[ igk[ ix ] - 1 ] - 1 ) * 2;

		// TODO: Copy from shared to global memory

	}
}

__global__ void kernel_init_psic( const  int * __restrict psic_index_nls, const  int * __restrict psic_index_nlsm, const  double * __restrict psi, double * psic, const int n, const int m, const int lda, const int ibnd )
{	   
	register int ix = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

	register int pos = ibnd * lda;
	register int pos_plus = (ibnd+1) * lda;

	register int psi_index = (ix + pos) * 2;
	register int psi_index_plus = (ix + pos_plus) * 2;

	if ( ix < n ) {

		// psic_index_nls[ix] = ( nls[ igk[ ix ] - 1 ] - 1 ) * 2;
		// psic_index_nlsm[ix] = ( nlsm[ igk[ ix ] - 1 ] - 1 ) * 2;

		if ( ibnd  < ( m - 1 ) ) {

			psic[ psic_index_nls[ix] ] = psi[ psi_index ] - psi[ psi_index_plus + 1 ];
			psic[ psic_index_nls[ix] + 1 ] = psi[ psi_index + 1 ] + psi[ psi_index_plus ];

			psic[ psic_index_nlsm[ix] ] = psi[ psi_index ] + psi[ psi_index_plus + 1 ];
			psic[ psic_index_nlsm[ix] + 1 ] = -1.0 * ( psi[ psi_index + 1 ] - psi[ psi_index_plus ] );

		} else {

			psic[ psic_index_nls[ix] ] = psi[ psi_index ];
			psic[ psic_index_nls[ix] + 1 ] = psi[ psi_index + 1 ];

			psic[ psic_index_nlsm[ix] ] = psi[ psi_index ];
			psic[ psic_index_nlsm[ix] + 1 ] = - 1.0 * psi[ psi_index + 1 ];

		}
	}
}

__global__ void kernel_save_hpsi(  const  int * __restrict psic_index_nls, const  int * __restrict psic_index_nlsm, double * hpsi, const  double * __restrict psic, const int n, const int m, const int lda, const int ibnd )
{	   
	register int ix = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

	register int pos = ibnd * lda;
	register int pos_plus = (ibnd+1) * lda;

	register int psi_index = (ix + pos) * 2;
	register int psi_index_plus = (ix + pos_plus) * 2;

	register double real_fp, cmplx_fp, real_fm, cmplx_fm;

	if ( ix < n ) {

		// psic_index_nls[ix] = (nls[ igk[ ix ] - 1 ] - 1) * 2;
		// psic_index_nlsm[ix] = (nlsm[ igk[ ix ] - 1 ] - 1) * 2;

		if( ibnd  < ( m - 1 ) ) {

			real_fp = ( psic[ psic_index_nls[ix] ] + psic[ psic_index_nlsm[ix] ] ) * 0.5;
			cmplx_fp = ( psic[ psic_index_nls[ix] + 1 ] + psic[ psic_index_nlsm[ix] + 1 ] ) * 0.5;

			real_fm = ( psic[ psic_index_nls[ix] ] - psic[ psic_index_nlsm[ix] ] ) * 0.5;
			cmplx_fm = ( psic[ psic_index_nls[ix] + 1 ] - psic[ psic_index_nlsm[ix] + 1 ] ) * 0.5;

			hpsi[ psi_index ] = hpsi[ psi_index ] + real_fp;
			hpsi[ psi_index + 1 ] = hpsi[ psi_index + 1 ] + cmplx_fm;

			hpsi[ psi_index_plus ] = hpsi[ psi_index_plus ] + cmplx_fp;
			hpsi[ psi_index_plus + 1 ] = hpsi[ psi_index_plus + 1 ] - real_fm;

		} else {

			hpsi[ psi_index ] = hpsi[ psi_index ] + psic[ psic_index_nls[ix] ];
			hpsi[ psi_index + 1 ] = hpsi[ psi_index + 1 ] + psic[ psic_index_nls[ix] + 1 ];

		}
	}
}


extern "C"  int vloc_psi_cuda_(int * ptr_lda, int * ptr_nrxxs, int * ptr_nr1s, int * ptr_nr2s, int * ptr_nr3s, int * ptr_n, int * ptr_m, void * psi, double * v, void * hpsi, int * igk, int * nls, int * nlsm, int * ptr_ngms, int * ptr_ngm)
{
//	hipfftHandle p_global;

    void * psic_D, * psi_D, * hpsi_D; // hipfftDoubleComplex*
	void * v_D; // double*
	void * igk_D, * nls_D, * nlsm_D, * psic_index_nls_D, *psic_index_nlsm_D; // int*
	int blocksPerGrid, ibnd;

	double tscale;

	int n = (* ptr_n);
	int m = (* ptr_m);
	int m_fake;
	int nr1s = (* ptr_nr1s);
	int nr2s = (* ptr_nr2s);
	int nr3s = (* ptr_nr3s);
	int nrxxs = (* ptr_nrxxs);
	int ngms = (* ptr_ngms);
	int ngm = (* ptr_ngm);
	int lda = (* ptr_lda);
#if defined(__CUDA_NOALLOC)
    int ierr;
#endif
    int size_psic = nr1s * nr2s * nr3s;

#if defined(__CUDA_DEBUG)
	printf("[CUDA DEBUG] VLOC_PSI_GAMMA] n=%d\n",n); fflush(stdout);
#endif

	/* Padding -- really necessary?*/
	if (m%2 == 0)
		m_fake = m ;
	else
		m_fake = m + 1;

	blocksPerGrid = ( n + __CUDA_TxB_VLOCPSI_PSIC__ - 1) / __CUDA_TxB_VLOCPSI_PSIC__ ;
	if ( blocksPerGrid > __CUDA_MAXNUMBLOCKS__) {
		fprintf( stderr, "\n[VLOC_PSI_GAMMA_OPT] kernel_init_psic cannot run, blocks requested ( %d ) > blocks allowed!!!", blocksPerGrid );
		return 1;
	}

	blocksPerGrid = ( (nrxxs * 2) + __CUDA_TxB_VLOCPSI_PROD__  - 1) / __CUDA_TxB_VLOCPSI_PROD__ ;
	if ( blocksPerGrid > __CUDA_MAXNUMBLOCKS__) {
		fprintf( stderr, "\n[VLOC_PSI_GAMMA_OPT] kernel_vec_prod cannot run, blocks requested ( %d ) > blocks allowed!!!", blocksPerGrid );
		return 1;
	}

	blocksPerGrid = ( n + __CUDA_TxB_VLOCPSI_HPSI__ - 1) / __CUDA_TxB_VLOCPSI_HPSI__ ;
	if ( blocksPerGrid > __CUDA_MAXNUMBLOCKS__) {
		fprintf( stderr, "\n[VLOC_PSI_GAMMA_OPT] kernel_save_hpsi cannot run, blocks requested ( %d ) > blocks allowed!!!", blocksPerGrid );
		return 1;
	}

	hipSetDevice(qe_gpu_bonded[0]);

#if defined(__CUDA_NOALLOC)
	/* Do real allocation */
	ierr = hipMalloc ( (void**) &(qe_dev_scratch[0]), (size_t) qe_gpu_mem_unused[0] );
    qecudaGenericErr((hipError_t) ierr, "VLOC_PSI_GAMMA_OPT", "error memory allocation (qe_dev_scratch)");

#if defined(__CUDA_KERNEL_MEMSET)
	qecudaSafeCall( hipMemset( qe_dev_scratch[0], 0, (size_t) qe_gpu_mem_unused[0] ) );
#endif
#endif

	size_t shift = 0;
	psic_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( size_psic )*sizeof( hipfftDoubleComplex );
	psi_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( lda * m_fake )*sizeof( hipfftDoubleComplex );
	hpsi_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( lda * m_fake )*sizeof( hipfftDoubleComplex );
	v_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( nrxxs )*sizeof( double );
	nls_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (ngms % 2 == 0)? ngms : ngms + 1 )*sizeof(int);
	nlsm_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (ngm % 2 == 0)? ngm : ngm + 1 )*sizeof(int);
	igk_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (ngm % 2 == 0)? ngm : ngm + 1 )*sizeof(int);
	psic_index_nls_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (n % 2 == 0)? n : n + 1 )*sizeof(int);
	psic_index_nlsm_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (n % 2 == 0)? n : n + 1 )*sizeof(int);

	// now shift contains the amount of byte required on the GPU to compute
	if ( shift > qe_gpu_mem_unused[0] ) {
		fprintf( stderr, "\n[VLOC_PSI_GAMMA] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", shift, qe_gpu_mem_unused[0] );
#if defined(__CUDA_NOALLOC)
		/* Deallocating... */
		ierr = hipFree ( qe_dev_scratch[0] );
	    qecudaGenericErr((hipError_t) ierr, "VLOC_PSI_GAMMA_OPT", "error memory deallocation (qe_dev_scratch)");
#endif
		return 1;
	}

#if defined(__CUDA_KERNEL_MEMSET)
	qecudaSafeCall( hipMemset( psi_D, 0, sizeof( hipfftDoubleComplex ) * lda * m_fake ) );
	qecudaSafeCall( hipMemset( psic_index_nls_D , 0, sizeof( int ) * n ) );
	qecudaSafeCall( hipMemset( psic_index_nlsm_D , 0, sizeof( int ) * n ) );
#endif
	qecudaSafeCall( hipMemcpy( nls_D, nls,  sizeof( int ) * ngms, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( nlsm_D, nlsm,  sizeof( int ) * ngm, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( igk_D, igk,  sizeof( int ) * n, hipMemcpyHostToDevice ) );


	blocksPerGrid = ( n + __CUDA_TxB_VLOCPSI_BUILD_PSIC__ - 1) / __CUDA_TxB_VLOCPSI_BUILD_PSIC__ ;
	dim3 dimGrid(blocksPerGrid);
	dim3 dimBlock(__CUDA_TxB_VLOCPSI_BUILD_PSIC__);
	build_psic_index<<< dimGrid, dimBlock >>>( (int *) nls_D, (int *) nlsm_D, (int *) igk_D, (int *) psic_index_nls_D, (int *) psic_index_nlsm_D, n );
	qecudaGetLastError("kernel launch failure");

#if defined(__CUDA_DEBUG)
	printf("[VLOC_PSI_GAMMA_OPT] psic_index_nls_D psic_index_nlsm_D computed\n"); fflush(stdout);
#endif

	qecudaSafeCall( hipMemcpy( psi_D, psi,  sizeof( hipfftDoubleComplex ) * lda * m, hipMemcpyHostToDevice ) );
#if defined(__CUDA_KERNEL_MEMSET)
	if (m_fake > m) {
		qecudaSafeCall( hipMemset( (psi_D + ( lda * m )) , 0, sizeof( hipfftDoubleComplex ) * size_psic ) ); // Post-set of (m_fake) zeros
	}
#endif
	qecudaSafeCall( hipMemcpy( hpsi_D, hpsi,  sizeof( hipfftDoubleComplex ) * lda * m, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( v_D, v,  sizeof( double ) * nrxxs, hipMemcpyHostToDevice ) );

//	qecheck_cufft_call( hipfftPlan3d( &p_global, nr3s, nr2s,  nr1s, HIPFFT_Z2Z ) );
    qecheck_cufft_call( hipfftSetStream(qeCudaFFT_dffts,qecudaStreams[ 0 ]) );

	for( ibnd =  0; ibnd < m_fake; ibnd = ibnd + 2 )
	{
		qecudaSafeCall( hipMemset( psic_D , 0, size_psic * sizeof( hipfftDoubleComplex ) ) );

		blocksPerGrid = ( n + __CUDA_TxB_VLOCPSI_PSIC__ - 1) / __CUDA_TxB_VLOCPSI_PSIC__ ;
		kernel_init_psic<<<blocksPerGrid, __CUDA_TxB_VLOCPSI_PSIC__ >>>( (int *) psic_index_nls_D, (int *) psic_index_nlsm_D, (double *) psi_D, (double *) psic_D, n, m, lda, ibnd );
		qecudaGetLastError("kernel launch failure");

		qecheck_cufft_call( hipfftExecZ2Z( qeCudaFFT_dffts, (hipfftDoubleComplex *) psic_D, (hipfftDoubleComplex *) psic_D, HIPFFT_BACKWARD ) );

		blocksPerGrid = ( (nrxxs * 2) + __CUDA_TxB_VLOCPSI_PROD__  - 1) / __CUDA_TxB_VLOCPSI_PROD__ ;
		kernel_vec_prod<<<blocksPerGrid, __CUDA_TxB_VLOCPSI_PROD__ >>>( (double *) psic_D, (double *) v_D , nrxxs );
		qecudaGetLastError("kernel launch failure");

		qecheck_cufft_call( hipfftExecZ2Z( qeCudaFFT_dffts, (hipfftDoubleComplex *) psic_D, (hipfftDoubleComplex *) psic_D, HIPFFT_FORWARD ) );

		tscale = 1.0 / (double) ( size_psic );
		hipblasZdscal(qecudaHandles[ 0 ] , size_psic, &tscale, (hipfftDoubleComplex *) psic_D, 1);

		blocksPerGrid = ( n + __CUDA_TxB_VLOCPSI_HPSI__ - 1) / __CUDA_TxB_VLOCPSI_HPSI__ ;
		kernel_save_hpsi<<<blocksPerGrid, __CUDA_TxB_VLOCPSI_HPSI__ >>>( (int *) psic_index_nls_D, (int *) psic_index_nlsm_D, (double *) hpsi_D, (double *) psic_D, n, m, lda, ibnd );
		qecudaGetLastError("kernel launch failure");

	}

	qecudaSafeCall( hipMemcpy( hpsi, (hipfftDoubleComplex *) hpsi_D, sizeof( hipfftDoubleComplex ) * lda * m, hipMemcpyDeviceToHost ) );

//	qecheck_cufft_call( hipfftDestroy(p_global) );

#if defined(__CUDA_NOALLOC)
	/* Deallocating... */
	ierr = hipFree ( qe_dev_scratch[0] );
    qecudaGenericErr((hipError_t) ierr, "VLOC_PSI_GAMMA_OPT", "error memory allocation (qe_dev_scratch)");
#else
#if defined(__CUDA_KERNEL_MEMSET)
	qecudaSafeCall( hipMemset( qe_dev_scratch[0], 0, (size_t) qe_gpu_mem_unused[0] ) );
#endif
#endif

	return 0;
}

#if defined(__CUDA_MULTIPLAN_FFT)
extern "C" void vloc_psi_multiplan_cuda_(int * ptr_lda, int * ptr_nrxxs, int * ptr_nr1s, int * ptr_nr2s, int * ptr_nr3s, int * ptr_n, int * ptr_m, void * psi, double * v, void * hpsi, int * igk, int * nls, int * nlsm, int * ptr_ngms, int * ptr_ngm)
{
	hipfftHandle p_global;

	void * psic_D, * psi_D, * hpsi_D; // hipfftDoubleComplex *
	void * v_D; // double *
	void * igk_D, * nls_D, * nlsm_D; // int *

	double tscale;

	int n = (* ptr_n);
	int m = (* ptr_m);
	int nr1s = (* ptr_nr1s);
	int nr2s = (* ptr_nr2s);
	int nr3s = (* ptr_nr3s);
	int nrxxs = (* ptr_nrxxs);
	int ngms = (* ptr_ngms);
	int ngm = (* ptr_ngm);
	int lda = (* ptr_lda);
	int size_psic = nr1s * nr2s * nr3s;
    int ierr;
	int array[3];
	int dim_multiplepsic, n_singlepsic, n_multiplepsic, size_multiplepsic, v_size;
	int m_fake, m_buf, blocksPerGrid, i, j;

	psic_D = (hipfftDoubleComplex * ) qe_dev_scratch[0];

	/* Padding */
	if (m%2 == 0) {
		m_buf = m;
		m_fake = m_buf/2 ;
	} else {
		m_buf = m+1;
		m_fake = m_buf/2;
	}

	dim_multiplepsic = __NUM_FFT_MULTIPLAN__;
	n_multiplepsic = m_fake/dim_multiplepsic;
	n_singlepsic = m_fake%dim_multiplepsic;

	size_multiplepsic = size_psic * (dim_multiplepsic);

	hipSetDevice(qe_gpu_bonded[0]);

	size_t shift = 0;
	psic_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( size_multiplepsic )*sizeof( hipfftDoubleComplex );
	psi_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( n * m_buf )*sizeof( hipfftDoubleComplex );
	hpsi_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( n * m_buf )*sizeof( hipfftDoubleComplex );
	v_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( nrxxs )*sizeof( double );
	nls_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (ngms % 2 == 0)? ngms : ngms + 1 )*sizeof(int);
	nlsm_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (ngm % 2 == 0)? ngm : ngm + 1 )*sizeof(int);
	igk_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (n % 2 == 0)? n : n + 1 )*sizeof(int);

	// now	shift contains the amount of byte required on the GPU to compute

	if ( shift > qe_gpu_mem_unused[0] ) {
		fprintf( stderr, "\n[VLOC_PSI_GAMMA] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", shift, qe_gpu_mem_unused[0] );
		exit(EXIT_FAILURE);
	}

	qecudaSafeCall( hipMemset( qe_dev_scratch[0], 0, (size_t) qe_gpu_mem_unused[0] ) );

	qecudaSafeCall( hipMemcpy( psi_D, psi,  sizeof( hipfftDoubleComplex ) * n * m_buf, hipMemcpyHostToDevice ) );
	shift = ( n * m )*sizeof( hipfftDoubleComplex );
	qecudaSafeCall( hipMemset( (hipfftDoubleComplex*)( (char*)psi_D + shift) , 0, sizeof( hipfftDoubleComplex ) * m_buf ) ); // Post-set of (m_fake) zeros

	qecudaSafeCall( hipMemcpy( hpsi_D, hpsi,  sizeof( hipfftDoubleComplex ) * n * m, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( nls_D, nls,  sizeof( int ) * ngms, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( nlsm_D, nlsm,  sizeof( int ) * ngm, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( igk_D, igk,  sizeof( int ) * n, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( v_D, v,  sizeof( double ) * nrxxs, hipMemcpyHostToDevice ) );

	array[0] = nr3s;
	array[1] = nr2s;
	array[2] = nr1s;

	v_size = nr1s * nr2s * nr3s;

	if ( n_multiplepsic > 0 ) {

		qecheck_cufft_call( hipfftPlanMany( &p_global, 3, array, NULL, 1, 0, NULL,1,0,HIPFFT_Z2Z,dim_multiplepsic ) );
        qecheck_cufft_call( hipfftSetStream(p_global,qecudaStreams[ 0 ]) );

		for(j = 0; j< (m_fake-n_singlepsic); j+=dim_multiplepsic ) {

			qecudaSafeCall( hipMemset( psic_D , 0, dim_multiplepsic * size_psic * sizeof( hipfftDoubleComplex ) ) );

			blocksPerGrid = ( ( n * 2) + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
			for (i = 0; i < dim_multiplepsic; i++  )
			{
				shift = 2*i*size_psic*sizeof(double);
				kernel_init_psic<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (int*) nls_D, (int*) nlsm_D, (int*) igk_D, (double*) psi_D, (double*) ( (char*) psic_D + shift), n, m, lda, ((j+i)*2) );
				qecudaGetLastError("kernel launch failure");
			}

			qecheck_cufft_call( hipfftExecZ2Z( p_global,  (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D , HIPFFT_BACKWARD ) );

			blocksPerGrid = ( (v_size * 2) + __CUDA_THREADPERBLOCK__  - 1) / __CUDA_THREADPERBLOCK__ ;
			for( i = 0; i < dim_multiplepsic; i++ ) {
				shift = 2*i*size_psic*sizeof(double);
				kernel_vec_prod<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (double*) ( (char*)psic_D + shift), (double*) v_D , v_size );
				qecudaGetLastError("kernel launch failure");
			}

			qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D, HIPFFT_FORWARD ) );

			tscale = 1.0 / (double) ( size_psic );
			hipblasZdscal(qecudaHandles[ 0 ] , size_psic*dim_multiplepsic, &tscale, (hipDoubleComplex *) psic_D, 1);

			blocksPerGrid = ( ( n * 2) + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
			for (i = 0; i < dim_multiplepsic; i++  )
			{
				shift = 2*i*size_psic*sizeof(double);
				kernel_save_hpsi<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (int*) nls_D, (int*) nlsm_D, (int*) igk_D, (double*) hpsi_D, (double*) ( (char*) psic_D + shift), n, m, lda, ((j+i)*2) );
				qecudaGetLastError("kernel launch failure");
			}
		}

		qecheck_cufft_call( hipfftDestroy(p_global) );
	}

	if (n_singlepsic > 0 ) {

		qecudaSafeCall( hipMemset( psic_D , 0, n_singlepsic * size_psic * sizeof( hipfftDoubleComplex ) ) );

		qecheck_cufft_call( hipfftPlanMany( &p_global, 3, array, NULL, 1, 0, NULL,1,0,HIPFFT_Z2Z,n_singlepsic ) );
        qecheck_cufft_call( hipfftSetStream(p_global,qecudaStreams[ 0 ]) );

		blocksPerGrid = ( ( n * 2) + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
		for (i = 0; i < n_singlepsic; i++  )
		{
			shift = 2*i*size_psic*sizeof(double);
			kernel_init_psic<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (int*) nls_D, (int*) nlsm_D, (int*) igk_D, (double*) psi_D, (double*) ( (char*) psic_D + shift), n, m, lda, (dim_multiplepsic*n_multiplepsic + i)*2 );
			qecudaGetLastError("kernel launch failure");
		}

		qecheck_cufft_call( hipfftExecZ2Z( p_global,  (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D , HIPFFT_BACKWARD ) );

		blocksPerGrid = ( (v_size * 2) + __CUDA_THREADPERBLOCK__  - 1) / __CUDA_THREADPERBLOCK__ ;
		for( i = 0; i < n_singlepsic; i++ ){
			shift = 2*i*size_psic*sizeof(double);
			kernel_vec_prod<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (double*) ((char*) psic_D + shift), (double*) v_D , v_size );
			qecudaGetLastError("kernel launch failure");
		}

		tscale = 1.0 / (double) ( size_psic );
		hipblasZdscal(qecudaHandles[ 0 ] , size_psic * n_singlepsic, &tscale, (hipDoubleComplex *) psic_D, 1);

		qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D, HIPFFT_FORWARD ) );

		blocksPerGrid = ( ( n * 2) + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
		for (i = 0; i < n_singlepsic; i++  )
		{
			shift = 2*i*size_psic*sizeof(double);
			kernel_save_hpsi<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (int*) nls_D, (int*) nlsm_D, (int*) igk_D, (double*) hpsi_D, (double*) ( (char*) psic_D + shift), n, m, lda, (dim_multiplepsic*n_multiplepsic + i)*2 );
			qecudaGetLastError("kernel launch failure");
		}

		qecheck_cufft_call( hipfftDestroy(p_global) );
	}

	qecudaSafeCall( hipMemcpy( hpsi, hpsi_D, sizeof( hipfftDoubleComplex ) * n * m, hipMemcpyDeviceToHost ) );
	qecudaSafeCall( hipMemset( qe_dev_scratch[0], 0, (size_t) qe_gpu_mem_unused[0] ) );

}
#endif
