#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2001-2013 Quantum ESPRESSO Foundation
 *
 * This file is distributed under the terms of the
 * GNU General Public License. See the file `License'
 * in the root directory of the present distribution,
 * or http://www.gnu.org/copyleft/gpl.txt .
 *
 */

#include <stdlib.h>
#include <stdio.h>

#include "cuda_env.h"

typedef double fftw_complex[2];

__global__ void kernel_vec_prod_k( double *a, const  double * __restrict b, int dimx )
{	   
	register int ix = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	register double sup;
	register int ii = ix / 2;

	if ( ix < ( dimx * 2 ) ) {
		sup = a[ix] * b[ii];
		a[ix] = sup;
	}

	return;
}

__global__ void kernel_init_psic_k( const  int * __restrict nls, const  int * __restrict igk, const  double * __restrict psi, double *psic, const int n, const int lda, const int ibnd )
{	   
	int ix = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	// int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int psic_index_nls, psi_index = ( ix + ( ibnd * lda ) ) * 2;

	if ( ix < n ) {
		psic_index_nls = ( nls[ igk[ ix ] - 1 ] - 1 ) * 2;
		psic[ psic_index_nls ] = psi[ psi_index ];
		psic[ psic_index_nls + 1 ] = psi[ psi_index + 1 ];
	}

	return;
}

__global__ void kernel_save_hpsi_k( const  int * __restrict nls, const  int * __restrict igk, double * hpsi, const  double * __restrict psic, const int n, const int ibnd )
{	   
	int ix = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	int pos = ibnd * n;
	int psic_index_nls, psi_index = (ix + pos) * 2;

	if ( ix < (n) ) {
		psic_index_nls = (nls[ igk[ ix ] - 1 ] - 1) * 2;
		hpsi[ psi_index ] = hpsi[ psi_index ] + psic[ psic_index_nls ];
		hpsi[ psi_index + 1 ] = hpsi[ psi_index + 1 ] + psic[ psic_index_nls + 1 ];
	}

	return;
}


extern "C" int vloc_psi_cuda_k_( int * ptr_lda, int * ptr_nrxxs, int * ptr_nr1s, int * ptr_nr2s, int * ptr_nr3s, int * ptr_n, int * ptr_m, hipfftDoubleComplex * psi, double * v, fftw_complex * hpsi, int * igk, int * nls, int * ptr_ngms)
{
	hipfftHandle p_global;
	fftw_complex * psic = NULL;

	void * psic_D, * psi_D; // hipfftDoubleComplex *
	void * v_D; // double *
	void * igk_D, * nls_D; // int*

	double tscale;

	int n = (* ptr_n);
	int m = (* ptr_m);
	int nr1s = (* ptr_nr1s);
	int nr2s = (* ptr_nr2s);
	int nr3s = (* ptr_nr3s);
	int nrxxs = (* ptr_nrxxs);
	int ngms = (* ptr_ngms);
	int lda = (* ptr_lda);
#if defined(__CUDA_NOALLOC)
	int ierr;
#endif
	int size_psic = nr1s * nr2s * nr3s;
	int j, ibnd;

	dim3 threads2_psic(__CUDA_TxB_VLOCPSI_PSIC__);
	dim3 grid2_psic( qe_compute_num_blocks(n, threads2_psic.x) );

	dim3 threads2_prod(__CUDA_TxB_VLOCPSI_PROD__);
	dim3 grid2_prod( qe_compute_num_blocks((nrxxs * 2), threads2_prod.x) );

#if defined(__CUDA_DEBUG)
	printf("[CUDA DEBUG] VLOC_PSI_K\n"); fflush(stdout);
#endif

	if ( grid2_psic.x > __CUDA_MAXNUMBLOCKS__) {
		fprintf( stderr, "\n[VLOC_PSI_K] kernel_init_psic_k cannot run, blocks requested ( %d ) > blocks allowed!!!", grid2_psic.x );
		return 1;
	}

	if ( grid2_prod.x > __CUDA_MAXNUMBLOCKS__) {
		fprintf( stderr, "\n[VLOC_PSI_K] kernel_vec_prod cannot run, blocks requested ( %d ) > blocks allowed!!!", grid2_prod.x );
		return 1;
	}

	hipSetDevice(qe_gpu_bonded[0]);

#if defined(__CUDA_NOALLOC)
	/* Do real allocation */
	ierr = hipMalloc ( (void**) &(qe_dev_scratch[0]), (size_t) qe_gpu_mem_unused[0] );
	qecudaGenericErr((hipError_t) ierr, "VLOC_PSI_K", "error in memory allocation (qe_dev_scratch)");
#endif

	size_t shift = 0;
	psic_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( size_psic )*sizeof( hipfftDoubleComplex );
	psi_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( lda * m )*sizeof( hipfftDoubleComplex );
	v_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( nrxxs )*sizeof( double );
	nls_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (ngms % 2 == 0)? ngms : ngms + 1 )*sizeof(int);
	igk_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (n % 2 == 0)? n : n + 1 )*sizeof(int);
	// now	shift contains the amount of byte required on the GPU to compute

	if ( shift > qe_gpu_mem_unused[0] ) {
		fprintf( stderr, "\n[VLOC_PSI_K] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", shift, qe_gpu_mem_unused[0] );
#if defined(__CUDA_NOALLOC)
		/* Deallocating... */
		ierr = hipFree ( qe_dev_scratch[0] );
		qecudaGenericErr((hipError_t) ierr, "VLOC_PSI_K", "error memory release (qe_dev_scratch)");
#endif
		return 1;
	}

	qecudaSafeCall( hipMemcpy( psi_D, psi,  sizeof( hipfftDoubleComplex ) * lda * m, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( v_D, v,  sizeof( double ) * nrxxs, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( nls_D, nls,  sizeof( int ) * ngms, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( igk_D, igk,  sizeof( int ) * n, hipMemcpyHostToDevice ) );

	qecheck_cufft_call( hipfftPlan3d( &p_global, nr3s, nr2s,  nr1s, HIPFFT_Z2Z ) );
	qecheck_cufft_call( hipfftSetStream(p_global,qecudaStreams[ 0 ]) );

	qecudaSafeCall( hipHostAlloc ( (void**) &psic, size_psic * sizeof( fftw_complex ), hipHostMallocPortable ) );

	for ( ibnd =  0; ibnd < m; ibnd = ibnd + 1) {

		hipDeviceSynchronize();

		qecudaSafeCall( hipMemset( psic_D, 0, size_psic * sizeof( hipfftDoubleComplex ) ) );

		kernel_init_psic_k<<< grid2_psic, threads2_psic, 0, qecudaStreams[ 0 ] >>>(
				(int *) nls_D, (int *) igk_D, (double *) psi_D, (double *) psic_D, n, lda, ibnd );
		qecudaGetLastError("kernel launch failure");

		qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex *) psic_D, (hipfftDoubleComplex *) psic_D, HIPFFT_BACKWARD ) );

		kernel_vec_prod_k<<< grid2_prod, threads2_prod, 0, qecudaStreams[ 0 ] >>>(
				(double *) psic_D, (double *) v_D , nrxxs );
		qecudaGetLastError("kernel launch failure");

		for ( j = 0; j <  n && ibnd > 0; j++ ) {
			hpsi[ j + ( ( ibnd  - 1 ) * lda ) ][0] += psic[ nls [ igk[ j ] - 1  ] - 1 ][0];
			hpsi[ j + ( ( ibnd  - 1 ) * lda ) ][1] += psic[ nls [ igk[ j ] - 1  ] - 1 ][1];
		}

		qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex *) psic_D,
				(hipfftDoubleComplex *)psic_D, HIPFFT_FORWARD ) );

		tscale = 1.0 / (double) ( size_psic );

		hipblasZdscal(qecudaHandles[ 0 ] , size_psic, &tscale, (hipDoubleComplex *) psic_D, 1);

		qecudaSafeCall( hipMemcpy( psic, psic_D, sizeof( hipfftDoubleComplex ) * size_psic, hipMemcpyDeviceToHost ) );

//	    for( j = 0; j <  n; j++ ) {
//	      hpsi[ j + ( ibnd * lda ) ][0] += psic[ nls [ igk[ j ] - 1  ] - 1 ][0];
//	      hpsi[ j + ( ibnd * lda ) ][1] += psic[ nls [ igk[ j ] - 1  ] - 1 ][1];
//	    }

	}

	for ( j = 0; j <  n; j++ ) {
		hpsi[ j + ( ( m - 1 ) * lda ) ][0] += psic[ nls [ igk[ j ] - 1  ] - 1 ][0];
		hpsi[ j + ( ( m - 1 ) * lda ) ][1] += psic[ nls [ igk[ j ] - 1  ] - 1 ][1];
	}

	qecheck_cufft_call( hipfftDestroy(p_global) );

#if defined(__CUDA_NOALLOC)
	/* Deallocating... */
	ierr = hipFree ( qe_dev_scratch[0] );
	qecudaGenericErr((hipError_t) ierr, "VLOC_PSI_K", "error memory release (qe_dev_scratch)");
#else

#if defined(__CUDA_KERNEL_MEMSET)
	qecudaSafeCall( hipMemset( qe_dev_scratch[0], 0, (size_t) qe_gpu_mem_unused[0] ) );
#endif

#endif

	return 0;
}

/* This method will be updated in the next build */
#if defined(__CUDA_MULTIPLAN_FFT) && !defined(__CUDA_NOALLOC) && !defined(__CUDA_PRELOAD)
extern "C" void vloc_psi_multiplan_cuda_k_(  int * ptr_lda, int * ptr_nrxxs, int * ptr_nr1s, int * ptr_nr2s, int * ptr_nr3s, int * ptr_n, int * ptr_m, hipfftDoubleComplex * psi, double * v, fftw_complex * hpsi, int * igk, int * nls, int * ptr_ngms)
{
	void * psic_D, * psi_D; // hipfftDoubleComplex *
	fftw_complex * psic = NULL;
	void * v_D; // double *
	void * igk_D, * nls_D; // int *

	double tscale;

	int n = (* ptr_n);
	int m = (* ptr_m);
	int nr1s = (* ptr_nr1s);
	int nr2s = (* ptr_nr2s);
	int nr3s = (* ptr_nr3s);
	int nrxxs = (* ptr_nrxxs);
	int ngms = (* ptr_ngms);
	int lda = (* ptr_lda);
    int ierr;
	int dim_multiplepsic, n_singlepsic, n_multiplepsic, size_multiplepsic, i, j, k;
	int array[3];
	int size_psic = nr1s * nr2s * nr3s;
	int last;

	size_t buffer_size = 0L;

	dim3 threads2_psic(__CUDA_TxB_VLOCPSI_PSIC__);
	dim3 grid2_psic( qe_compute_num_blocks(n, threads2_psic.x) );

	dim3 threads2_prod(__CUDA_TxB_VLOCPSI_PROD__);
	dim3 grid2_prod( qe_compute_num_blocks((nrxxs * 2), threads2_prod.x) );

	hipSetDevice(qe_gpu_bonded[0]);

	dim_multiplepsic = __NUM_FFT_MULTIPLAN__;

	n_multiplepsic = m/dim_multiplepsic;
	n_singlepsic = m%dim_multiplepsic;

	size_multiplepsic = size_psic * dim_multiplepsic;

	buffer_size = size_multiplepsic * sizeof( hipfftDoubleComplex ) + sizeof( hipfftDoubleComplex ) * n * m + sizeof( int ) * ngms + sizeof( int ) * n + sizeof( double ) * nrxxs;

	if ( buffer_size > qe_gpu_mem_unused[0] ) {
		fprintf( stderr, "\n[VLOC_PSI_K] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", buffer_size, qe_gpu_mem_unused[0] );
		exit(EXIT_FAILURE);
	}

	size_t shift = 0;
	psic_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( size_psic )*sizeof( hipfftDoubleComplex );
	psi_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( lda * m )*sizeof( hipfftDoubleComplex );
	v_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( nrxxs )*sizeof( double );
	nls_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (ngms % 2 == 0)? ngms : ngms + 1 )*sizeof(int);
	igk_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (n % 2 == 0)? n : n + 1 )*sizeof(int);

	// now	shift contains the amount of byte required on the GPU to compute

	if ( shift > qe_gpu_mem_unused[0] ) {
		fprintf( stderr, "\n[VLOC_PSI_K] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", shift, qe_gpu_mem_unused[0] );
		exit(EXIT_FAILURE);
	}

	qecudaSafeCall( hipMemcpy( nls_D, nls,  sizeof( int ) * ngms, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( igk_D, igk,  sizeof( int ) * n, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( psi_D, psi,  sizeof( hipfftDoubleComplex ) * n * m, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( v_D, v,  sizeof( double ) * nrxxs, hipMemcpyHostToDevice ) );

	array[0] = nr3s;
	array[1] = nr2s;
	array[2] = nr1s;

	hipfftHandle p_global;

	if ( n_multiplepsic > 0 ) {

		qecheck_cufft_call( hipfftPlanMany( &p_global, 3, array, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, dim_multiplepsic ) );
        qecheck_cufft_call( hipfftSetStream(p_global,qecudaStreams[ 0 ]));

		qecudaSafeCall( hipHostAlloc ( (void**) &psic, size_multiplepsic * sizeof( fftw_complex ), hipHostMallocPortable ) );

		for(j = 0; j< (m-n_singlepsic); j = j + dim_multiplepsic ) {

			qecudaSafeCall( hipMemset( psic_D, 0, size_psic * dim_multiplepsic * sizeof( hipfftDoubleComplex ) ) );

			for( i = 0; i < dim_multiplepsic; i++ ) {
				kernel_init_psic_k<<< grid2_psic, threads2_psic >>>( (int*) nls_D, (int*) igk_D, (double*) psi_D, (double*) psic_D, n, lda, (j+i));
				qecudaGetLastError("kernel launch failure");
			}

			qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D, HIPFFT_BACKWARD ) );

			for( i = 0; i < dim_multiplepsic; i++ ) {
				kernel_vec_prod_k<<< grid2_proc, threads2_proc >>>( (double*) psic_D, (double*) v_D , nrxxs );
				qecudaGetLastError("kernel launch failure");
			}

			for( i = 0; i < dim_multiplepsic; i++ ) {
				if ( (i+j) > 0 ) {
					for ( k = 0; k <  n ; k++ ) {
						hpsi[ k + ( ( (i+j)  - 1 ) * lda ) ][0] += psic[ nls [ igk[ k ] - 1  ] - 1 ][0];
						hpsi[ k + ( ( (i+j)  - 1 ) * lda ) ][1] += psic[ nls [ igk[ k ] - 1  ] - 1 ][1];
					}
				}
			}

			qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D, HIPFFT_FORWARD ) );

			tscale = 1.0 / (double) ( size_psic );
			hipblasZdscal(qecudaHandles[ 0 ] , size_psic*dim_multiplepsic, &tscale, (hipDoubleComplex *) psic_D, 1);

			qecudaSafeCall( hipMemcpy( psic, (hipfftDoubleComplex*) psic_D, sizeof( hipfftDoubleComplex ) * size_psic * dim_multiplepsic, hipMemcpyDeviceToHost ) );
		}

		last = dim_multiplepsic*n_multiplepsic;
		for ( k = 0; k <  n; k++ ) {
			hpsi[ k + ( ( last - 1 ) * lda ) ][0] += psic[ nls [ igk[ k ] - 1  ] - 1 ][0];
			hpsi[ k + ( ( last - 1 ) * lda ) ][1] += psic[ nls [ igk[ k ] - 1  ] - 1 ][1];
		}

		qecheck_cufft_call( hipfftDestroy(p_global) );

		qecudaSafeCall( hipMemset( psic_D, 0, size_psic * dim_multiplepsic * sizeof( hipfftDoubleComplex ) ) );
		qecudaSafeCall( hipHostFree(psic));

	}

	if (n_singlepsic > 0 ) {

		printf("n_singlepsic\n");fflush(stdout);

		qecheck_cufft_call( hipfftPlanMany( &p_global, 3, array, NULL, 1, 0, NULL, 1, 0,HIPFFT_Z2Z, n_singlepsic ) );
        qecheck_cufft_call( hipfftSetStream(p_global,qecudaStreams[ 0 ]) );

		qecudaSafeCall( hipHostAlloc ( (void**) &psic, n_singlepsic * size_psic * sizeof( hipfftDoubleComplex ), hipHostMallocPortable ) );
		qecudaSafeCall( hipMemset( psic_D, 0, n_singlepsic * size_psic * sizeof( hipfftDoubleComplex ) ) );

		for( i = 0; i < n_singlepsic; i++ ) {
			shift = 2*i*size_psic*sizeof(double);
			kernel_init_psic_k<<< grid2_psic, threads2_psic >>>( (int*) nls_D, (int*) igk_D, (double*) psi_D, (double*) ( (char*) psic_D + shift ), n, lda, ((dim_multiplepsic*n_multiplepsic) +i) );
			qecudaGetLastError("kernel launch failure");
		}

		qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D, HIPFFT_BACKWARD ) );

		for( i = 0; i < n_singlepsic; i++ ) {
			shift = 2*i*size_psic*sizeof(double);
			kernel_vec_prod_k<<< grid2_prod, threads2_prod >>>( (double *) ( (char*) psic_D + shift ), (double *) v_D , nrxxs );
			qecudaGetLastError("kernel launch failure");
		}

		qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D, HIPFFT_FORWARD ) );

		tscale = 1.0 / (double) ( size_psic );
		hipblasZdscal(qecudaHandles[ 0 ] , n_singlepsic * size_psic, &tscale, (hipDoubleComplex *) psic_D, 1);

		qecudaSafeCall( hipMemcpy( psic, psic_D, sizeof( hipfftDoubleComplex ) * n_singlepsic * size_psic, hipMemcpyDeviceToHost ) );

		for( i = 0; i < n_singlepsic; i++ ) {
			for( k = 0; k <  n; k++ ) {
				hpsi[ k + ( ((dim_multiplepsic*n_multiplepsic) + i) * lda ) ][0] += psic[ nls [ igk[ k ] - 1  ] - 1 ][0];
				hpsi[ k + ( ((dim_multiplepsic*n_multiplepsic) + i) * lda ) ][1] += psic[ nls [ igk[ k ] - 1  ] - 1 ][1];
			}
		}

		qecheck_cufft_call( hipfftDestroy(p_global) );
		qecudaSafeCall( hipHostFree(psic));
	}
}
#endif
