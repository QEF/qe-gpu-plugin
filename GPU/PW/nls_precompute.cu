#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2001-2013 Quantum ESPRESSO Foundation
 *
 * This file is distributed under the terms of the
 * GNU General Public License. See the file `License'
 * in the root directory of the present distribution,
 * or http://www.gnu.org/copyleft/gpl.txt .
 *
 */

/*    ***********************************    *
 *    * IT WORKS ONLY IF __CUDA_NOALLOC *    *
 *    ***********************************    * */

#include <stdlib.h>
#include <stdio.h>

#include "cuda_env.h"

__global__ void build_psic_gamma_index(const  int * __restrict nls, const  int * __restrict nlsm, const  int * __restrict igk, int * psic_index_nls, int * psic_index_nlsm, const int n ){

	register int ix = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

	// TODO: Fetch in shared memory igk[ ix ]
	// TODO: In-place index calculation

	if ( ix < n ) {
		psic_index_nls[ix] = ( nls[ igk[ ix ] - 1 ] - 1 ) * 2;
		psic_index_nlsm[ix] = ( nlsm[ igk[ ix ] - 1 ] - 1 ) * 2;
	}

	// TODO: Copy from shared to global memory
}

__global__ void build_psic_k_index(const  int * __restrict nls, const  int * __restrict igk, int * psic_index_nls, const int n ){

	register int ix = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

	// TODO: Fetch in shared memory igk[ ix ]
	// TODO: In-place index calculation

	if ( ix < n ) {
		psic_index_nls[ix] = ( nls[ igk[ ix ] - 1 ] - 1 ) * 2;
	}

	// TODO: Copy from shared to global memory

}

extern "C" int nls_precompute_k_( int * ptr_lda, int * ptr_n, int * igk, int * nls, int * ptr_ngms)
{
	void * igk_D, * nls_D; // int*

	int  blocksPerGrid;
	int n = (* ptr_n);
	int ngms = (* ptr_ngms);
	int lda = (* ptr_lda);
#if defined(__CUDA_NOALLOC)
	int ierr;
#endif
	size_t shift;

	hipStream_t  vlocStreams[ MAX_QE_GPUS ];
	hipblasHandle_t vlocHandles[ MAX_QE_GPUS ];

#if defined(__CUDA_DEBUG)
	printf("[NLS_PRECOMPUTE_K] n=%d\n",n); fflush(stdout);
#endif

	blocksPerGrid = ( n + __CUDA_TxB_VLOCPSI_BUILD_PSIC__ - 1) / __CUDA_TxB_VLOCPSI_BUILD_PSIC__ ;
	if ( blocksPerGrid > 65535) {
		fprintf( stderr, "\n[NLS_PRECOMPUTE_K] build_psic_k_index cannot run, blocks requested ( %d ) > blocks allowed!!!", blocksPerGrid );
		return 1;
	}

	hipSetDevice(qe_gpu_bonded[0]);

	// Have I already use preloaded_nls_D previously? Yes, then clean
	if (preloaded_nls_D != NULL){
		/* Deallocating... */
#if defined(__CUDA_DEBUG)
	    printf("[NLS_PRECOMPUTE_K] Detected previous index computation, deallocate before recompute  \n"); fflush(stdout);
#endif
		ierr = hipFree ( preloaded_nls_D );
		if(ierr != hipSuccess) {
			fprintf( stderr, "\nError in memory release, program will be terminated!!! Bye...\n\n" );
			exit(EXIT_FAILURE);
		}

		qe_gpu_mem_unused[0] = qe_gpu_mem_tot[0];
	}

	shift = ( (n % 2 == 0)? n : n + 1 )*sizeof(int);

	if ( shift > qe_gpu_mem_unused[0] ) {
		fprintf( stderr, "\n[NLS_PRECOMPUTE_K] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", shift, qe_gpu_mem_unused[0] );
		return 1;
	}

	ierr = hipMalloc ( (void**) &preloaded_nls_D, (size_t) n*sizeof(int) );
	if ( ierr != hipSuccess) {
		fprintf( stderr, "\nError in memory allocation, program will be terminated (%d)!!! Bye...\n\n", ierr );
		exit(EXIT_FAILURE);
	}

#if defined(__CUDA_DEBUG)
	printf("[NLS_PRECOMPUTE_K] preloaded_nls_D allocated (used = %lu byte)\n",(size_t) n*sizeof(int)); fflush(stdout);
#endif

#if defined(__CUDA_KERNEL_MEMSET)
	qecudaSafeCall( hipMemset( (int *) preloaded_nls_D, 0, (size_t) n*sizeof(int) ) );
#endif

	qe_gpu_mem_unused[0] = qe_gpu_mem_tot[0] - shift;

	/* Do real allocation */
	ierr = hipMalloc ( (void**) &(qe_dev_scratch[0]), (size_t) qe_gpu_mem_unused[0] );
	if ( ierr != hipSuccess) {
		fprintf( stderr, "*** NLS_PRECOMPUTE_K *** Error in memory allocation (qe_dev_scratch), program will be terminated (%d)!!! Bye...\n\n", ierr );
		exit(EXIT_FAILURE);
	}

#if defined(__CUDA_KERNEL_MEMSET)
	qecudaSafeCall( hipMemset( qe_dev_scratch[0], 0, (size_t) qe_gpu_mem_unused[0] ) );
#endif

	shift = 0;
	nls_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (ngms % 2 == 0)? ngms : ngms + 1 )*sizeof(int);
	igk_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (n % 2 == 0)? n : n + 1 )*sizeof(int);

#if defined(__CUDA_DEBUG)
	printf("[NLS_PRECOMPUTE_K] qe_dev_scratch allocated (used = %lu byte)\n", shift); fflush(stdout);
#endif

	if ( shift > qe_gpu_mem_unused[0] ) {

		fprintf( stderr, "\n[NLS_PRECOMPUTE_K] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", shift, qe_gpu_mem_unused[0] );

		/* Deallocating... */
		ierr = hipFree ( preloaded_nls_D );
		if(ierr != hipSuccess) {
			fprintf( stderr, "\n*** NLS_PRECOMPUTE_K *** Error in memory release (preloaded_nls_D), program will be terminated!!! Bye...\n\n" );
			exit(EXIT_FAILURE);
		}

		ierr = hipFree ( qe_dev_scratch[0] );
		if(ierr != hipSuccess) {
			fprintf( stderr, "\n*** NLS_PRECOMPUTE_K *** Error in memory release (qe_dev_scratch), program will be terminated!!! Bye...\n\n" );
			exit(EXIT_FAILURE);
		}
		return 1;
	}

	qecudaSafeCall( hipMemcpy( nls_D, nls,  sizeof( int ) * ngms, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( igk_D, igk,  sizeof( int ) * n, hipMemcpyHostToDevice ) );

	// blocksPerGrid = ( n + __CUDA_TxB_VLOCPSI_BUILD_PSIC__ - 1) / __CUDA_TxB_VLOCPSI_BUILD_PSIC__ ;
	build_psic_k_index<<<blocksPerGrid, __CUDA_TxB_VLOCPSI_BUILD_PSIC__ >>>( (int *) nls_D, (int *) igk_D, (int *) preloaded_nls_D, n );
	qecudaGetLastError("kernel launch failure");

#if defined(__CUDA_DEBUG)
	printf("[NLS_PRECOMPUTE_K] preloaded_nls_D populated\n"); fflush(stdout);
#endif

	/* Deallocating... but NOT preloaded_nls_D */
	ierr = hipFree ( qe_dev_scratch[0] );
	if(ierr != hipSuccess) {
		fprintf( stderr, "\nError in memory release (qe_dev_scratch), program will be terminated!!! Bye...\n\n" );
		exit(EXIT_FAILURE);
	}

	// guard
	hipDeviceSynchronize();

#if defined(__CUDA_DEBUG)
	printf("[NLS_PRECOMPUTE_K] qe_dev_scratch cleaned\n"); fflush(stdout);
#endif

	return 0;
}

extern "C" int nls_precompute_gamma_( int * ptr_lda, int * ptr_n, int * igk, int * nls,  int * nlsm, int * ptr_ngms, int * ptr_ngm)
{
	void * igk_D, * nls_D , * nlsm_D ; // int*

	int  blocksPerGrid;
	int n = (* ptr_n);
	int ngms = (* ptr_ngms);
	int ngm = (* ptr_ngm);
	int lda = (* ptr_lda);
#if defined(__CUDA_NOALLOC)
	int ierr;
#endif
	size_t shift;

	hipStream_t  vlocStreams[ MAX_QE_GPUS ];
	hipblasHandle_t vlocHandles[ MAX_QE_GPUS ];

#if defined(__CUDA_DEBUG)
	printf("[NLS_PRECOMPUTE_GAMMA] n=%d\n",n); fflush(stdout);
#endif

	blocksPerGrid = ( n + __CUDA_TxB_VLOCPSI_BUILD_PSIC__ - 1) / __CUDA_TxB_VLOCPSI_BUILD_PSIC__ ;
	if ( blocksPerGrid > 65535) {
		fprintf( stderr, "\n[NLS_PRECOMPUTE_GAMMA] build_psic_gamma_index cannot run, blocks requested ( %d ) > blocks allowed!!!", blocksPerGrid );
		return 1;
	}

	hipSetDevice(qe_gpu_bonded[0]);

	/*
	 * ASSUMPTION: preloaded_nls_D and preloaded_nlsm_D works always in pair
	 *             in gamma calculation
	 */

	// Have I already use preloaded_nls_D previously? Yes, then clean
	if (preloaded_nls_D != NULL){
		/* Deallocating... */
#if defined(__CUDA_DEBUG)
	    printf("[NLS_PRECOMPUTE_GAMMA] Detected previous index computation, deallocate before recompute  \n"); fflush(stdout);
#endif
		ierr = hipFree ( preloaded_nls_D );
		if(ierr != hipSuccess) {
			fprintf( stderr, "\nError in memory release, program will be terminated!!! Bye...\n\n" );
			exit(EXIT_FAILURE);
		}

		ierr = hipFree ( preloaded_nlsm_D );
		if(ierr != hipSuccess) {
			fprintf( stderr, "\nError in memory release, program will be terminated!!! Bye...\n\n" );
			exit(EXIT_FAILURE);
		}

		qe_gpu_mem_unused[0] = qe_gpu_mem_tot[0];
	}

	shift = ( (n % 2 == 0)? n : n + 1 )*sizeof(int)*2;

	if ( shift > qe_gpu_mem_unused[0] ) {
		fprintf( stderr, "\n[NLS_PRECOMPUTE_K] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", shift, qe_gpu_mem_unused[0] );
		return 1;
	}

	ierr = hipMalloc ( (void**) &preloaded_nls_D, (size_t) n*sizeof(int) );
	if ( ierr != hipSuccess) {
		fprintf( stderr, "\nError in memory allocation, program will be terminated (%d)!!! Bye...\n\n", ierr );
		exit(EXIT_FAILURE);
	}

#if defined(__CUDA_DEBUG)
	printf("[NLS_PRECOMPUTE_GAMMA] preloaded_nls_D allocated (used = %lu byte)\n",(size_t) n*sizeof(int)); fflush(stdout);
#endif

	ierr = hipMalloc ( (void**) &preloaded_nlsm_D, (size_t) n*sizeof(int) );
	if ( ierr != hipSuccess) {
		fprintf( stderr, "\nError in memory allocation, program will be terminated (%d)!!! Bye...\n\n", ierr );
		exit(EXIT_FAILURE);
	}

#if defined(__CUDA_DEBUG)
	printf("[NLS_PRECOMPUTE_GAMMA] preloaded_nlsm_D allocated (used = %lu byte)\n",(size_t) n*sizeof(int)); fflush(stdout);
#endif

#if defined(__CUDA_KERNEL_MEMSET)
	qecudaSafeCall( hipMemset( (int *) preloaded_nls_D, 0, (size_t) n*sizeof(int) ) );
	qecudaSafeCall( hipMemset( (int *) preloaded_nlsm_D, 0, (size_t) n*sizeof(int) ) );
#endif

	qe_gpu_mem_unused[0] = qe_gpu_mem_tot[0] - shift;

	/* Do real allocation */
	ierr = hipMalloc ( (void**) &(qe_dev_scratch[0]), (size_t) qe_gpu_mem_unused[0] );
	if ( ierr != hipSuccess) {
		fprintf( stderr, "*** NLS_PRECOMPUTE_GAMMA *** Error in memory allocation (qe_dev_scratch), program will be terminated (%d)!!! Bye...\n\n", ierr );
		exit(EXIT_FAILURE);
	}

#if defined(__CUDA_KERNEL_MEMSET)
	qecudaSafeCall( hipMemset( qe_dev_scratch[0], 0, (size_t) qe_gpu_mem_unused[0] ) );
#endif

	shift = 0;
	nls_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (ngms % 2 == 0)? ngms : ngms + 1 )*sizeof(int);
	nlsm_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (ngm % 2 == 0)? ngm : ngm + 1 )*sizeof(int);
	igk_D = (char*) qe_dev_scratch[0] + shift;
	shift += ( (n % 2 == 0)? n : n + 1 )*sizeof(int);

#if defined(__CUDA_DEBUG)
	printf("[NLS_PRECOMPUTE_GAMMA] qe_dev_scratch allocated (used = %lu byte)\n", shift); fflush(stdout);
#endif

	if ( shift > qe_gpu_mem_unused[0] ) {

		fprintf( stderr, "\n[NLS_PRECOMPUTE_GAMMA] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", shift, qe_gpu_mem_unused[0] );

		/* Deallocating... */
		ierr = hipFree ( preloaded_nls_D );
		if(ierr != hipSuccess) {
			fprintf( stderr, "\n*** NLS_PRECOMPUTE_GAMMA *** Error in memory release (preloaded_nls_D), program will be terminated!!! Bye...\n\n" );
			exit(EXIT_FAILURE);
		}

		/* Deallocating... */
		ierr = hipFree ( preloaded_nlsm_D );
		if(ierr != hipSuccess) {
			fprintf( stderr, "\n*** NLS_PRECOMPUTE_GAMMA *** Error in memory release (preloaded_nlsm_D), program will be terminated!!! Bye...\n\n" );
			exit(EXIT_FAILURE);
		}

		ierr = hipFree ( qe_dev_scratch[0] );
		if(ierr != hipSuccess) {
			fprintf( stderr, "\n*** NLS_PRECOMPUTE_GAMMA *** Error in memory release (qe_dev_scratch), program will be terminated!!! Bye...\n\n" );
			exit(EXIT_FAILURE);
		}
		return 1;
	}

	qecudaSafeCall( hipMemcpy( nls_D, nls,  sizeof( int ) * ngms, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( nlsm_D, nlsm,  sizeof( int ) * ngm, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( igk_D, igk,  sizeof( int ) * n, hipMemcpyHostToDevice ) );

	// blocksPerGrid = ( n + __CUDA_TxB_VLOCPSI_BUILD_PSIC__ - 1) / __CUDA_TxB_VLOCPSI_BUILD_PSIC__ ;
	build_psic_gamma_index<<<blocksPerGrid, __CUDA_TxB_VLOCPSI_BUILD_PSIC__ >>>( (int *) nls_D, (int *) nlsm_D, (int *) igk_D, (int *) preloaded_nls_D, (int *) preloaded_nlsm_D, n );
	qecudaGetLastError("kernel launch failure");

#if defined(__CUDA_DEBUG)
	printf("[NLS_PRECOMPUTE_K] preloaded_nls_D populated\n"); fflush(stdout);
#endif

	/* Deallocating... but NOT preloaded_nls_D */
	ierr = hipFree ( qe_dev_scratch[0] );
	if(ierr != hipSuccess) {
		fprintf( stderr, "\nError in memory release (qe_dev_scratch), program will be terminated!!! Bye...\n\n" );
		exit(EXIT_FAILURE);
	}

	// guard
	hipDeviceSynchronize();

#if defined(__CUDA_DEBUG)
	printf("[NLS_PRECOMPUTE_K] qe_dev_scratch cleaned\n"); fflush(stdout);
#endif

	return 0;
}


extern "C" int nls_precompute_k_cleanup_( )
{
	int ierr;

	/* Deallocating preloaded_nls_D */
	ierr = hipFree ( preloaded_nls_D );
	if(ierr != hipSuccess) {
		fprintf( stderr, "\n*** NLS_PRECOMPUTE_K *** Error in memory release (preloaded_nls_D), program will be terminated!!! Bye...\n\n" );
		exit(EXIT_FAILURE);
	}

	preloaded_nls_D = NULL;
	qe_gpu_mem_unused[0] = qe_gpu_mem_tot[0];
	// qe_dev_scratch[0] = qe_dev_zero_scratch[0];

#if defined(__CUDA_DEBUG)
	printf("[NLS_PRECOMPUTE_K_CLEANUP] preloaded_nls_D cleaned \n"); fflush(stdout);
#endif

	return 0;
}

extern "C" int nls_precompute_gamma_cleanup_( )
{
	int ierr;

	/* Deallocating preloaded_nls_D */
	ierr = hipFree ( preloaded_nls_D );
	if(ierr != hipSuccess) {
		fprintf( stderr, "\n*** NLS_PRECOMPUTE_GAMMA_CLEANUP *** Error in memory release (preloaded_nls_D), program will be terminated!!! Bye...\n\n" );
		exit(EXIT_FAILURE);
	}

	preloaded_nls_D = NULL;

#if defined(__CUDA_DEBUG)
	printf("[NLS_PRECOMPUTE_GAMMA_CLEANUP] preloaded_nls_D cleaned \n"); fflush(stdout);
#endif

	/* Deallocating preloaded_nlsm_D */
	ierr = hipFree ( preloaded_nlsm_D );
	if(ierr != hipSuccess) {
		fprintf( stderr, "\n*** NLS_PRECOMPUTE_GAMMA_CLEANUP *** Error in memory release (preloaded_nlsm_D), program will be terminated!!! Bye...\n\n" );
		exit(EXIT_FAILURE);
	}

	preloaded_nls_D = NULL;

#if defined(__CUDA_DEBUG)
	printf("[NLS_PRECOMPUTE_GAMMA_CLEANUP] preloaded_nlsm_D cleaned \n"); fflush(stdout);
#endif

	qe_gpu_mem_unused[0] = qe_gpu_mem_tot[0];
	// qe_dev_scratch[0] = qe_dev_zero_scratch[0];

	return 0;
}



