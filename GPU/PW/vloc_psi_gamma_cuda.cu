#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2001-2012 Quantum ESPRESSO group
 * Copyright (C) 2010-2011 Irish Centre for High-End Computing (ICHEC)
 *
 * This file is distributed under the terms of the
 * GNU General Public License. See the file `License'
 * in the root directory of the present distribution,
 * or http://www.gnu.org/copyleft/gpl.txt .
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include "cuda_env.h"

extern "C" void start_clock_(char * label, unsigned int length_arg );
extern "C" void stop_clock_(char * label, unsigned int length_arg );

__global__ void kernel_vec_prod( double *a, const  double * __restrict b, int dimx )
{	   
	register int ix = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	register double sup;
	register int ii = ix / 2;

	if( ix < ( dimx * 2 ) ){
		sup = a[ix] * b[ii];
		a[ix] = sup;
	}
}

__global__ void kernel_init_psic( const  int * __restrict nls, const  int * __restrict nlsm, const  int * __restrict igk, const  double * __restrict psi, double * psic, const int n, const int m, const int lda, const int ibnd )
{	   
	register int ix = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	register int pos = ibnd * lda;
	register int pos_plus = (ibnd+1) * lda;

	register int psic_index_nls;
	register int psic_index_nlsm;

	register int psi_index = (ix + pos) * 2;
	register int psi_index_plus = (ix + pos_plus) * 2;

	if ( ix < n ) {

		psic_index_nls = ( nls[ igk[ ix ] - 1 ] - 1 ) * 2;
		psic_index_nlsm = ( nlsm[ igk[ ix ] - 1 ] - 1 ) * 2;

		if ( ibnd  < ( m - 1 ) ) {

			psic[ psic_index_nls ] = psi[ psi_index ] - psi[ psi_index_plus + 1 ];
			psic[ psic_index_nls + 1 ] = psi[ psi_index + 1 ] + psi[ psi_index_plus ];

			psic[ psic_index_nlsm ] = psi[ psi_index ] + psi[ psi_index_plus + 1 ];
			psic[ psic_index_nlsm + 1 ] = -1.0 * ( psi[ psi_index + 1 ] - psi[ psi_index_plus ] );

		} else {

			psic[ psic_index_nls ] = psi[ psi_index ];
			psic[ psic_index_nls + 1 ] = psi[ psi_index + 1 ];

			psic[ psic_index_nlsm ] = psi[ psi_index ];
			psic[ psic_index_nlsm + 1 ] = - 1.0 * psi[ psi_index + 1 ];

		}
	}
}

__global__ void kernel_save_hpsi( const  int * __restrict nls, const  int * __restrict nlsm, const  int * __restrict igk, double * hpsi, const  double * __restrict psic, const int n, const int m, const int lda, const int ibnd )
{	   
	register int ix = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	register int pos = ibnd * lda;
	register int pos_plus = (ibnd+1) * lda;

	register int psic_index_nls;
	register int psic_index_nlsm;

	register int psi_index = (ix + pos) * 2;
	register int psi_index_plus = (ix + pos_plus) * 2;

	register double real_fp, cmplx_fp, real_fm, cmplx_fm;

	if ( ix < n ) {

		psic_index_nls = (nls[ igk[ ix ] - 1 ] - 1) * 2;
		psic_index_nlsm = (nlsm[ igk[ ix ] - 1 ] - 1) * 2;

		if( ibnd  < ( m - 1 ) ) {

			real_fp = ( psic[ psic_index_nls ] + psic[ psic_index_nlsm ] ) * 0.5;
			cmplx_fp = ( psic[ psic_index_nls + 1 ] + psic[ psic_index_nlsm + 1 ] ) * 0.5;

			real_fm = ( psic[ psic_index_nls ] - psic[ psic_index_nlsm ] ) * 0.5;
			cmplx_fm = ( psic[ psic_index_nls + 1 ] - psic[ psic_index_nlsm + 1 ] ) * 0.5;

			hpsi[ psi_index ] = hpsi[ psi_index ] + real_fp;
			hpsi[ psi_index + 1 ] = hpsi[ psi_index + 1 ] + cmplx_fm;

			hpsi[ psi_index_plus ] = hpsi[ psi_index_plus ] + cmplx_fp;
			hpsi[ psi_index_plus + 1 ] = hpsi[ psi_index_plus + 1 ] - real_fm;

		} else {

			hpsi[ psi_index ] = hpsi[ psi_index ] + psic[ psic_index_nls ];
			hpsi[ psi_index + 1 ] = hpsi[ psi_index + 1 ] + psic[ psic_index_nls + 1 ];

		}
	}
}


extern "C"  int vloc_psi_cuda_(int * ptr_lda, int * ptr_nrxxs, int * ptr_nr1s, int * ptr_nr2s, int * ptr_nr3s, int * ptr_n, int * ptr_m, void * psi, double * v, void * hpsi, int * igk, int * nls, int * nlsm, int * ptr_ngms, int * ptr_ngm)
{
	hipfftHandle p_global;

    void * psic_D, * psi_D, * hpsi_D; // hipfftDoubleComplex*
	void * v_D; // double*
	void * local_igk_D, * local_nls_D, * local_nlsm_D; // int*
	int blocksPerGrid, ibnd;

	double tscale;

	int n = (* ptr_n);
	int m = (* ptr_m);
	int m_fake;
	int nr1s = (* ptr_nr1s);
	int nr2s = (* ptr_nr2s);
	int nr3s = (* ptr_nr3s);
	int nrxxs = (* ptr_nrxxs);
	int ngms = (* ptr_ngms);
	int ngm = (* ptr_ngm);
	int lda = (* ptr_lda);

	int size_psic = nr1s * nr2s * nr3s;

	hipStream_t  vlocStreams[ MAX_QE_GPUS ];
	hipblasHandle_t vlocHandles[ MAX_QE_GPUS ];

//	size_t buffer_size = 0L;

#if defined(__CUDA_DEBUG)
	printf("\n[CUDA_DEBUG - VLOC_PSI_GAMMA] m = %d, n = %d, nrxxs = %d, size_psic = %d\n", m, n, nrxxs, size_psic);
	fflush(stdout);
#endif

	/* Padding -- really necessary?*/
	if (m%2 == 0)
		m_fake = m ;
	else
		m_fake = m + 1;

	hipSetDevice(qe_gpu_bonded[0]);

	if ( hipblasCreate( &vlocHandles[ 0 ] ) != HIPBLAS_STATUS_SUCCESS ) {
		printf("\n*** CUDA VLOC_PSI_GAMMA *** ERROR *** cublasInit() for device %d failed!",qe_gpu_bonded[0]);
		fflush(stdout);
		exit(EXIT_FAILURE);
	}

	if( hipStreamCreate( &vlocStreams[ 0 ] ) != hipSuccess ) {
		printf("\n*** CUDA VLOC_PSI_GAMMA *** ERROR *** creating stream for device %d failed!",qe_gpu_bonded[0]);
		fflush(stdout);
		exit(EXIT_FAILURE);
	}

	// qecudaSafeCall( hipMemset( dev_scratch_QE[0], 0, (size_t) cuda_memory_allocated[0] ) );

//	buffer_size = size_psic * sizeof( hipfftDoubleComplex ) + sizeof( hipfftDoubleComplex ) * lda * m_fake * 2 + sizeof( int ) * ngms + sizeof( int ) * ngm + sizeof( int ) * lda + sizeof( double ) * nrxxs;
//
//	if ( buffer_size > cuda_memory_unused[0] ) {
//		fprintf( stderr, "\n[VLOC_PSI_GAMMA] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", buffer_size, cuda_memory_allocated[0] );
//        return 1;
//	}

	blocksPerGrid = ( ( n * 2) + __CUDA_TxB_VLOCPSI_PSIC__ - 1) / __CUDA_TxB_VLOCPSI_PSIC__ ;
	if ( blocksPerGrid > 65535) {
		fprintf( stderr, "\n[VLOC_PSI_GAMMA] kernel_init_psic cannot run, blocks requested ( %d ) > blocks allowed!!!", blocksPerGrid );
		return 1;
	}

	blocksPerGrid = ( (nrxxs * 2) + __CUDA_TxB_VLOCPSI_PROD__  - 1) / __CUDA_TxB_VLOCPSI_PROD__ ;
	if ( blocksPerGrid > 65535) {
		fprintf( stderr, "\n[VLOC_PSI_GAMMA] kernel_vec_prod cannot run, blocks requested ( %d ) > blocks allowed!!!", blocksPerGrid );
		return 1;
	}

	blocksPerGrid = ( ( n * 2) + __CUDA_TxB_VLOCPSI_HPSI__ - 1) / __CUDA_TxB_VLOCPSI_HPSI__ ;
	if ( blocksPerGrid > 65535) {
		fprintf( stderr, "\n[VLOC_PSI_GAMMA] kernel_save_hpsi cannot run, blocks requested ( %d ) > blocks allowed!!!", blocksPerGrid );
		return 1;
	}

	size_t shift = 0;
	psic_D = (char*) dev_scratch_QE[0] + shift;
	shift += ( size_psic )*sizeof( hipfftDoubleComplex );
	psi_D = (char*) dev_scratch_QE[0] + shift;
	shift += ( lda * m_fake )*sizeof( hipfftDoubleComplex );
	hpsi_D = (char*) dev_scratch_QE[0] + shift;
	shift += ( lda * m_fake )*sizeof( hipfftDoubleComplex );
	v_D = (char*) dev_scratch_QE[0] + shift;
	shift += ( nrxxs )*sizeof( double );
#if defined(__CUDA_PRELOADING_DATA)
	// now	shift contains the amount of byte required on the GPU to compute
	local_nls_D = (void *) preloaded_nls_D;
	local_nlsm_D = (void *) preloaded_nlsm_D;
	local_igk_D = (void *) preloaded_igk_D;
#else
	local_nls_D = (char*) dev_scratch_QE[0] + shift;
	shift += ( (ngms % 2 == 0)? ngms : ngms + 1 )*sizeof(int);
	local_nlsm_D = (char*) dev_scratch_QE[0] + shift;
	shift += ( (ngm % 2 == 0)? ngm : ngm + 1 )*sizeof(int);
	local_igk_D = (char*) dev_scratch_QE[0] + shift;
	shift += ( (ngm % 2 == 0)? ngm : ngm + 1 )*sizeof(int);
#endif
	// now	shift contains the amount of byte required on the GPU to compute

	if ( shift > cuda_memory_unused[0] ) {
		fprintf( stderr, "\n[VLOC_PSI_GAMMA] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", shift, cuda_memory_allocated[0] );
        return 1;
	}

	// Before do anything force sync to terminate async data transfer
#if defined(__CUDA_PRELOADING_DATA) && defined(__CUDA_PRELOAD_PINNED)
	hipDeviceSynchronize();
#endif

	qecudaSafeCall( hipMemset( psi_D, 0, sizeof( hipfftDoubleComplex ) * lda * m_fake ) );
	qecudaSafeCall( hipMemcpy( psi_D, psi,  sizeof( hipfftDoubleComplex ) * lda * m, hipMemcpyHostToDevice ) );
	// ???
//	if (m_fake > m) {
//		qecudaSafeCall( hipMemset( (psi_D + ( lda * m )) , 0, sizeof( hipfftDoubleComplex ) * size_psic ) ); // Post-set of (m_fake) zeros
//	}

#if !defined(__CUDA_PRELOADING_DATA)
	qecudaSafeCall( hipMemcpy( local_nls_D, nls,  sizeof( int ) * ngms, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( local_nlsm_D, nlsm,  sizeof( int ) * ngm, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( local_igk_D, igk,  sizeof( int ) * n, hipMemcpyHostToDevice ) );
#endif
	qecudaSafeCall( hipMemcpy( hpsi_D, hpsi,  sizeof( hipfftDoubleComplex ) * lda * m, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( v_D, v,  sizeof( double ) * nrxxs, hipMemcpyHostToDevice ) );

	qecheck_cufft_call( hipfftPlan3d( &p_global, nr3s, nr2s,  nr1s, HIPFFT_Z2Z ) );

	if( hipfftSetStream(p_global,vlocStreams[ 0 ]) != HIPFFT_SUCCESS ) {
		printf("\n*** CUDA VLOC_PSI_GAMMA *** ERROR *** hipfftSetStream for device %d failed!",qe_gpu_bonded[0]);
		fflush(stdout);
		exit( EXIT_FAILURE );
	}

	for( ibnd =  0; ibnd < m_fake; ibnd = ibnd + 2 )
	{
		qecudaSafeCall( hipMemset( psic_D , 0, size_psic * sizeof( hipfftDoubleComplex ) ) );

		blocksPerGrid = ( ( n * 2) + __CUDA_TxB_VLOCPSI_PSIC__ - 1) / __CUDA_TxB_VLOCPSI_PSIC__ ;
		kernel_init_psic<<<blocksPerGrid, __CUDA_TxB_VLOCPSI_PSIC__ >>>( (int *) local_nls_D, (int *) local_nlsm_D, (int *) local_igk_D, (double *) psi_D, (double *) psic_D, n, m, lda, ibnd );
		qecudaGetLastError("kernel launch failure");

		qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex *) psic_D, (hipfftDoubleComplex *) psic_D, HIPFFT_BACKWARD ) );

		blocksPerGrid = ( (nrxxs * 2) + __CUDA_TxB_VLOCPSI_PROD__  - 1) / __CUDA_TxB_VLOCPSI_PROD__ ;
		kernel_vec_prod<<<blocksPerGrid, __CUDA_TxB_VLOCPSI_PROD__ >>>( (double *) psic_D, (double *) v_D , nrxxs );
		qecudaGetLastError("kernel launch failure");

		qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex *) psic_D, (hipfftDoubleComplex *) psic_D, HIPFFT_FORWARD ) );

		tscale = 1.0 / (double) ( size_psic );
		hipblasZdscal(vlocHandles[ 0 ] , size_psic, &tscale, (hipfftDoubleComplex *) psic_D, 1);

		blocksPerGrid = ( ( n * 2) + __CUDA_TxB_VLOCPSI_HPSI__ - 1) / __CUDA_TxB_VLOCPSI_HPSI__ ;
		kernel_save_hpsi<<<blocksPerGrid, __CUDA_TxB_VLOCPSI_HPSI__ >>>( (int *) local_nls_D, (int *) local_nlsm_D, (int *) local_igk_D, (double *) hpsi_D, (double *) psic_D, n, m, lda, ibnd );
		qecudaGetLastError("kernel launch failure");

	}

	qecudaSafeCall( hipMemcpy( hpsi, (hipfftDoubleComplex *) hpsi_D, sizeof( hipfftDoubleComplex ) * lda * m, hipMemcpyDeviceToHost ) );

	qecheck_cufft_call( hipfftDestroy(p_global) );

#if defined(__CUDA_KERNEL_MEMSET)
	qecudaSafeCall( hipMemset( dev_scratch_QE[0], 0, (size_t) cuda_memory_unused[0] ) );
#endif

	hipStreamDestroy( vlocStreams[ 0 ] );
	hipblasDestroy( vlocHandles[ 0 ]);

	return 0;
}


extern "C" void vloc_psi_multiplan_cuda_(int * ptr_lda, int * ptr_nrxxs, int * ptr_nr1s, int * ptr_nr2s, int * ptr_nr3s, int * ptr_n, int * ptr_m, void * psi, double * v, void * hpsi, int * igk, int * nls, int * nlsm, int * ptr_ngms, int * ptr_ngm)
{
	hipfftHandle p_global;

	void * psic_D, * psi_D, * hpsi_D; // hipfftDoubleComplex *
	void * v_D; // double *
	void * igk_D, * nls_D, * nlsm_D; // int *

	double tscale;

	int n = (* ptr_n);
	int m = (* ptr_m);
	int nr1s = (* ptr_nr1s);
	int nr2s = (* ptr_nr2s);
	int nr3s = (* ptr_nr3s);
	int nrxxs = (* ptr_nrxxs);
	int ngms = (* ptr_ngms);
	int ngm = (* ptr_ngm);
	int lda = (* ptr_lda);
	int size_psic = nr1s * nr2s * nr3s;

	int array[3];
	int dim_multiplepsic, n_singlepsic, n_multiplepsic, size_multiplepsic, v_size;
	int m_fake, m_buf, blocksPerGrid, i, j;

	hipStream_t  vlocStreams[ MAX_QE_GPUS ];
	hipblasHandle_t vlocHandles[ MAX_QE_GPUS ];

	psic_D = (hipfftDoubleComplex * ) dev_scratch_QE[0];

	/* Padding */
	if (m%2 == 0) {
		m_buf = m;
		m_fake = m_buf/2 ;
	} else {
		m_buf = m+1;
		m_fake = m_buf/2;
	}

	dim_multiplepsic = __NUM_FFT_MULTIPLAN__;
	n_multiplepsic = m_fake/dim_multiplepsic;
	n_singlepsic = m_fake%dim_multiplepsic;

	size_multiplepsic = size_psic * (dim_multiplepsic);

	hipSetDevice(qe_gpu_bonded[0]);

	if ( hipblasCreate( &vlocHandles[ 0 ] ) != HIPBLAS_STATUS_SUCCESS ) {
		printf("\n*** CUDA VLOC_PSI_GAMMA *** ERROR *** cublasInit() for device %d failed!",qe_gpu_bonded[0]);
		fflush(stdout);
		exit(EXIT_FAILURE);
	}

	if( hipStreamCreate( &vlocStreams[ 0 ] ) != hipSuccess ) {
		printf("\n*** CUDA VLOC_PSI_GAMMA *** ERROR *** creating stream for device %d failed!",qe_gpu_bonded[0]);
		fflush(stdout);
		exit(EXIT_FAILURE);
	}

	size_t shift = 0;
	psic_D = (char*) dev_scratch_QE[0] + shift;
	shift += ( size_multiplepsic )*sizeof( hipfftDoubleComplex );
	psi_D = (char*) dev_scratch_QE[0] + shift;
	shift += ( n * m_buf )*sizeof( hipfftDoubleComplex );
	hpsi_D = (char*) dev_scratch_QE[0] + shift;
	shift += ( n * m_buf )*sizeof( hipfftDoubleComplex );
	v_D = (char*) dev_scratch_QE[0] + shift;
	shift += ( nrxxs )*sizeof( double );
	nls_D = (char*) dev_scratch_QE[0] + shift;
	shift += ( (ngms % 2 == 0)? ngms : ngms + 1 )*sizeof(int);
	nlsm_D = (char*) dev_scratch_QE[0] + shift;
	shift += ( (ngm % 2 == 0)? ngm : ngm + 1 )*sizeof(int);
	igk_D = (char*) dev_scratch_QE[0] + shift;
	shift += ( (n % 2 == 0)? n : n + 1 )*sizeof(int);

	// now	shift contains the amount of byte required on the GPU to compute

	if ( shift > cuda_memory_allocated[0] ) {
		fprintf( stderr, "\n[VLOC_PSI_GAMMA] Problem don't fit in GPU memory --- memory requested ( %lu ) > memory allocated  (%lu )!!!", shift, cuda_memory_allocated[0] );
		exit(EXIT_FAILURE);
	}

	qecudaSafeCall( hipMemset( dev_scratch_QE[0], 0, (size_t) cuda_memory_allocated[0] ) );

	qecudaSafeCall( hipMemcpy( psi_D, psi,  sizeof( hipfftDoubleComplex ) * n * m_buf, hipMemcpyHostToDevice ) );
	shift = ( n * m )*sizeof( hipfftDoubleComplex );
	qecudaSafeCall( hipMemset( (hipfftDoubleComplex*)( (char*)psi_D + shift) , 0, sizeof( hipfftDoubleComplex ) * m_buf ) ); // Post-set of (m_fake) zeros

	qecudaSafeCall( hipMemcpy( hpsi_D, hpsi,  sizeof( hipfftDoubleComplex ) * n * m, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( nls_D, nls,  sizeof( int ) * ngms, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( nlsm_D, nlsm,  sizeof( int ) * ngm, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( igk_D, igk,  sizeof( int ) * n, hipMemcpyHostToDevice ) );
	qecudaSafeCall( hipMemcpy( v_D, v,  sizeof( double ) * nrxxs, hipMemcpyHostToDevice ) );

	array[0] = nr3s;
	array[1] = nr2s;
	array[2] = nr1s;

	v_size = nr1s * nr2s * nr3s;

	if ( n_multiplepsic > 0 ) {

		qecheck_cufft_call( hipfftPlanMany( &p_global, 3, array, NULL, 1, 0, NULL,1,0,HIPFFT_Z2Z,dim_multiplepsic ) );

		if( hipfftSetStream(p_global,vlocStreams[ 0 ]) != HIPFFT_SUCCESS ) {
			printf("\n*** CUDA VLOC_PSI_GAMMA *** ERROR *** hipfftSetStream for device %d failed!",qe_gpu_bonded[0]);
			fflush(stdout);
			exit( EXIT_FAILURE );
		}

		for(j = 0; j< (m_fake-n_singlepsic); j+=dim_multiplepsic ) {

			qecudaSafeCall( hipMemset( psic_D , 0, dim_multiplepsic * size_psic * sizeof( hipfftDoubleComplex ) ) );

			blocksPerGrid = ( ( n * 2) + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
			for (i = 0; i < dim_multiplepsic; i++  )
			{
				shift = 2*i*size_psic*sizeof(double);
				kernel_init_psic<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (int*) nls_D, (int*) nlsm_D, (int*) igk_D, (double*) psi_D, (double*) ( (char*) psic_D + shift), n, m, lda, ((j+i)*2) );
				qecudaGetLastError("kernel launch failure");
			}

			qecheck_cufft_call( hipfftExecZ2Z( p_global,  (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D , HIPFFT_BACKWARD ) );

			blocksPerGrid = ( (v_size * 2) + __CUDA_THREADPERBLOCK__  - 1) / __CUDA_THREADPERBLOCK__ ;
			for( i = 0; i < dim_multiplepsic; i++ ) {
				shift = 2*i*size_psic*sizeof(double);
				kernel_vec_prod<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (double*) ( (char*)psic_D + shift), (double*) v_D , v_size );
				qecudaGetLastError("kernel launch failure");
			}

			qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D, HIPFFT_FORWARD ) );

			tscale = 1.0 / (double) ( size_psic );
			hipblasZdscal(vlocHandles[ 0 ] , size_psic*dim_multiplepsic, &tscale, (hipDoubleComplex *) psic_D, 1);

			blocksPerGrid = ( ( n * 2) + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
			for (i = 0; i < dim_multiplepsic; i++  )
			{
				shift = 2*i*size_psic*sizeof(double);
				kernel_save_hpsi<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (int*) nls_D, (int*) nlsm_D, (int*) igk_D, (double*) hpsi_D, (double*) ( (char*) psic_D + shift), n, m, lda, ((j+i)*2) );
				qecudaGetLastError("kernel launch failure");
			}
		}

		qecheck_cufft_call( hipfftDestroy(p_global) );
	}

	if (n_singlepsic > 0 ) {

		qecudaSafeCall( hipMemset( psic_D , 0, n_singlepsic * size_psic * sizeof( hipfftDoubleComplex ) ) );

		qecheck_cufft_call( hipfftPlanMany( &p_global, 3, array, NULL, 1, 0, NULL,1,0,HIPFFT_Z2Z,n_singlepsic ) );

		if( hipfftSetStream(p_global,vlocStreams[ 0 ]) != HIPFFT_SUCCESS ) {
			printf("\n*** CUDA VLOC_PSI_GAMMA *** ERROR *** hipfftSetStream for device %d failed!",qe_gpu_bonded[0]);
			fflush(stdout);
			exit( EXIT_FAILURE );
		}

		blocksPerGrid = ( ( n * 2) + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
		for (i = 0; i < n_singlepsic; i++  )
		{
			shift = 2*i*size_psic*sizeof(double);
			kernel_init_psic<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (int*) nls_D, (int*) nlsm_D, (int*) igk_D, (double*) psi_D, (double*) ( (char*) psic_D + shift), n, m, lda, (dim_multiplepsic*n_multiplepsic + i)*2 );
			qecudaGetLastError("kernel launch failure");
		}

		qecheck_cufft_call( hipfftExecZ2Z( p_global,  (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D , HIPFFT_BACKWARD ) );

		blocksPerGrid = ( (v_size * 2) + __CUDA_THREADPERBLOCK__  - 1) / __CUDA_THREADPERBLOCK__ ;
		for( i = 0; i < n_singlepsic; i++ ){
			shift = 2*i*size_psic*sizeof(double);
			kernel_vec_prod<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (double*) ((char*) psic_D + shift), (double*) v_D , v_size );
			qecudaGetLastError("kernel launch failure");
		}

		tscale = 1.0 / (double) ( size_psic );
		hipblasZdscal(vlocHandles[ 0 ] , size_psic * n_singlepsic, &tscale, (hipDoubleComplex *) psic_D, 1);

		qecheck_cufft_call( hipfftExecZ2Z( p_global, (hipfftDoubleComplex*) psic_D, (hipfftDoubleComplex*) psic_D, HIPFFT_FORWARD ) );

		blocksPerGrid = ( ( n * 2) + __CUDA_THREADPERBLOCK__ - 1) / __CUDA_THREADPERBLOCK__ ;
		for (i = 0; i < n_singlepsic; i++  )
		{
			shift = 2*i*size_psic*sizeof(double);
			kernel_save_hpsi<<<blocksPerGrid, __CUDA_THREADPERBLOCK__ >>>( (int*) nls_D, (int*) nlsm_D, (int*) igk_D, (double*) hpsi_D, (double*) ( (char*) psic_D + shift), n, m, lda, (dim_multiplepsic*n_multiplepsic + i)*2 );
			qecudaGetLastError("kernel launch failure");
		}

		qecheck_cufft_call( hipfftDestroy(p_global) );
	}

	qecudaSafeCall( hipMemcpy( hpsi, hpsi_D, sizeof( hipfftDoubleComplex ) * n * m, hipMemcpyDeviceToHost ) );
	qecudaSafeCall( hipMemset( dev_scratch_QE[0], 0, (size_t) cuda_memory_allocated[0] ) );

	hipStreamDestroy( vlocStreams[ 0 ] );
	hipblasDestroy( vlocHandles[ 0 ]);
}
