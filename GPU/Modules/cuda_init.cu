#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2001-2012 Quantum ESPRESSO group
 * Copyright (C) 2010-2011 Irish Centre for High-End Computing (ICHEC)
 *
 * This file is distributed under the terms of the
 * GNU General Public License. See the file `License'
 * in the root directory of the present distribution,
 * or http://www.gnu.org/copyleft/gpl.txt .
 *
 */

#include <stdlib.h>
#include <stdio.h>

#include <hip/driver_types.h>

#if defined(__TIMELOG)
#include <time.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#endif

#include "cuda_env.h"

#if defined(__PHIGEMM)
#include "phigemm.h"
#endif

qeCudaMemDevPtr dev_scratch_QE;
qeCudaMemDevPtr dev_heap_QE;
qeCudaMemSizes cuda_memory_allocated;
qeCudaMemSizes device_memory_left;
qeCudaMemSizes device_memory_shift;
qeCudaMemSizes cuda_memory_unused;
qeCudaDevicesBond qe_gpu_bonded;

// global useful information
long ngpus_detected;
long ngpus_used;
long ngpus_per_process;
long procs_per_gpu;

// Pre-loaded data-structures
void * preloaded_eigts1_D = 0, * preloaded_eigts2_D = 0, * preloaded_eigts3_D = 0;
void * preloaded_ig1_D = 0, * preloaded_ig2_D = 0, * preloaded_ig3_D = 0;
void * preloaded_nlsm_D = 0, * preloaded_nls_D = 0, * preloaded_igk_D = 0;
short int preloaded_igk_flag;

extern "C" void paralleldetect_(int * lRankThisNode_ptr, int * lSizeThisNode_ptr , int * lRank_ptr);
extern "C" void mybarrier_();

#if defined(__TIMELOG)
double cuda_cclock(void)
{
	struct timeval tv;
	struct timezone tz;
	double t;

	gettimeofday(&tv, &tz);

	t = (double)tv.tv_sec;
	t += ((double)tv.tv_usec)/1000000.0;

	return t;
}
#endif


void gpubinding_(int lRankThisNode, int lSizeThisNode, int lRank){

	int lNumDevicesThisNode = 0;
	int i;

#if defined(__PARA)

	/* Attach all MPI processes on this node to the available GPUs
	 * in round-robin fashion
	 */
	hipGetDeviceCount(&lNumDevicesThisNode);

	if (lNumDevicesThisNode == 0 && lRankThisNode == 0)
	{
		printf("***ERROR: no CUDA-capable devices were found.\n");
//		MPI_Abort( MPI_COMM_WORLD, EXIT_FAILURE );
		exit(EXIT_FAILURE);
	}

	ngpus_detected = lNumDevicesThisNode;

	if ( (lSizeThisNode % lNumDevicesThisNode ) != 0  )
	{
		printf("***WARNING: unbalanced configuration (%d MPI per node, %d GPUs per node)\n", lSizeThisNode, lNumDevicesThisNode);
		fflush(stdout);
	}

	if (ngpus_detected <= lSizeThisNode ){
		/* if GPUs are less then (or equal of) the number of  MPI processes on a single node,
		 * then PWscf uses all the GPU and one single GPU is assigned to one or multiple MPI processes with overlapping. */
		ngpus_used = ngpus_detected;
		ngpus_per_process = 1;
	} else {
		/* multi-GPU in parallel calculations is allowed ONLY if CUDA >= 4.0 */

		/* if GPUs are more than the MPI processes on a single node,
		 * then PWscf uses all the GPU and one or more GPUs are assigned
		 * to every single MPI processes without overlapping.
		 * *** NOT IMPLEMENTED YET ***
		 */
		ngpus_used = ngpus_detected;
		ngpus_per_process = 1;
	}

	procs_per_gpu = (lSizeThisNode < lNumDevicesThisNode) ? lSizeThisNode : lSizeThisNode / lNumDevicesThisNode;

	for (i = 0; i < ngpus_per_process; i++) {

		qe_gpu_bonded[i] = lRankThisNode % lNumDevicesThisNode;

#if defined(__CUDA_DEBUG)
		printf("Binding GPU %d on node of rank: %d (internal rank:%d)\n", qe_gpu_bonded[i], lRank, lRankThisNode); fflush(stdout);
#endif

	}

#else

	hipGetDeviceCount(&lNumDevicesThisNode);

	if (lNumDevicesThisNode == 0)
	{
		fprintf( stderr,"***ERROR*** no CUDA-capable devices were found on the machine.\n");
		exit(EXIT_FAILURE);
	}

	ngpus_detected = lNumDevicesThisNode;

	/* multi-GPU in serial calculations is allowed ONLY if CUDA >= 4.0 */
#if defined(__MULTI_GPU)
	ngpus_used = ngpus_per_process = lNumDevicesThisNode;
#else
	ngpus_used = ngpus_per_process = 1;
#endif

	for (i = 0; i < ngpus_per_process; i++) {
		/* NOTE: qe_gpu_bonded[0] is ALWAYS the main device for non multi-GPU
		 *       kernels.
		 */
		qe_gpu_bonded[i] = i;
	}

	// NULL tag for phiGEMM
	lRank = -1;

#endif
}

#if defined(__PHIGEMM)
extern "C" void initphigemm_(int lRank){
	/* Compatibility with CUDA 4.x (latest phiGEMM): */

#if defined(__PHIGEMM_NOALLOC)
	phiGemmInit(ngpus_per_process , NULL, (qeCudaMemSizes*)&cuda_memory_allocated, (int *)qe_gpu_bonded, lRank);
#else
	phiGemmInit(ngpus_per_process , (qeCudaMemDevPtr*)&dev_scratch_QE, (qeCudaMemSizes*)&cuda_memory_allocated, (int *)qe_gpu_bonded, lRank);
#endif
}
#endif

extern "C" void preallocatedevicememory_(int lRank){

	int ierr = 0;
	int i;

	size_t free, total;

	for (i = 0; i < ngpus_per_process; i++) {

		/* query the real free memory, taking into account the "stack" */
		if ( hipSetDevice(qe_gpu_bonded[i]) != hipSuccess) {
			printf("*** ERROR *** hipSetDevice(%d) failed!", qe_gpu_bonded[i] ); fflush(stdout);
			exit(EXIT_FAILURE);
		}

		cuda_memory_allocated[i] = (size_t) 0;

		ierr = hipMalloc ( (void**) &(dev_scratch_QE[i]), cuda_memory_allocated[i] );
		if ( ierr != hipSuccess) {
			fprintf( stderr, "\nError in (first zero) memory allocation , program will be terminated!!! Bye...\n\n");
			exit(EXIT_FAILURE);
		}

#if defined(__PARA)
	}

	// MPI_Barrier(MPI_COMM_WORLD);
	mybarrier_();

	for (i = 0; i < ngpus_per_process; i++) {
#endif

		// see cuda_env.h for a description of the hack
		// this does *NOT* work if everything is not performed at the beginning...
#if defined(__CUDA_GET_MEM_HACK)
		free = (size_t)  __GPU_MEM_AMOUNT_HACK__;
#else
		hipMemGetInfo((size_t*)&free,(size_t*)&total);
#endif

#if defined(__CUDA_DEBUG)
#if defined(__PARA)
		printf("[GPU %d - rank: %d] before: %lu (total: %lu)\n", qe_gpu_bonded[i], lRank, (unsigned long)free, (unsigned long)total); fflush(stdout);
#else
		printf("[GPU %d] before: %lu (total: %lu)\n", qe_gpu_bonded[i], (unsigned long)free, (unsigned long)total); fflush(stdout);
#endif
#endif

#if defined(__PARA)
		cuda_memory_allocated[i] = (size_t) ((((free * __SCALING_MEM_FACTOR__ ) * 16.0) / 16.0) / procs_per_gpu);
		cuda_memory_unused[i] = cuda_memory_allocated[i];
#else
		cuda_memory_allocated[i] = (size_t) (((free * __SCALING_MEM_FACTOR__ ) * 16.0) / 16.0);
		cuda_memory_unused[i] = cuda_memory_allocated[i];
#endif


		// Temporary hack...
#if !defined(__PHIGEMM_NOALLOC)
		/* Do real allocation */
		ierr = hipMalloc ( (void**) &(dev_scratch_QE[i]), (size_t) cuda_memory_allocated[i] );
		if ( ierr != hipSuccess) {
			fprintf( stderr, "\nError in memory allocation, program will be terminated (%d)!!! Bye...\n\n", ierr );
			exit(EXIT_FAILURE);
		}
#endif

#if defined(__PARA)
	}

	// MPI_Barrier(MPI_COMM_WORLD);
	mybarrier_();

	for (i = 0; i < ngpus_per_process; i++) {
#endif

		/* It can be useful to track this information... */
#if defined(__CUDA_GET_MEM_HACK)
		free = __GPU_MEM_AMOUNT_HACK__ - cuda_memory_allocated[i];
#else
		hipMemGetInfo((size_t*)&free,(size_t*)&total);
#endif

		device_memory_left[i] = free;

		dev_heap_QE[i] = (char * ) dev_scratch_QE[i] + (32*(cuda_memory_allocated[i]/32));
		device_memory_shift[i] = 0;

#if defined(__CUDA_DEBUG)
#if defined(__PARA)
		printf("[GPU %d - rank: %d] after: %lu (total: %lu)\n", qe_gpu_bonded[i], lRank, (unsigned long)free, (unsigned long)total); fflush(stdout);
#else
		printf("[GPU %d] after: %lu (total: %lu)\n", qe_gpu_bonded[i], (unsigned long)free, (unsigned long)total); fflush(stdout);
#endif
#endif
	}
	
	// Print information on screen
#if defined(__PARA)
	if (lRank == 0) {	
#endif
	printf("\n"); fflush(stdout);
	printf("     *******************************************************************\n\n"); fflush(stdout);
#if defined(__PHIGEMM_HACK_CPUONLY)
	printf("       CPU-version plus call-by-call GEMM profiling"); fflush(stdout);
#else

	printf("       GPU-accelerated Quantum ESPRESSO \n\n"); fflush(stdout);

#if defined(__PARA)
	printf("       parallel      : yes (GPUs per node = %d, GPUs per process = %d)  \n", ngpus_detected, ngpus_per_process); fflush(stdout);
#else
	printf("       parallel      : no (GPUs detected = %d, GPUs used = %d)  \n", ngpus_detected, ngpus_used); fflush(stdout);
#endif

#if defined(__OPENACC)
	printf("       OpenACC       : yes \n"); fflush(stdout);
#endif

#if defined(__CUDA_PINNED)
    printf("       pinned memory : yes \n"); fflush(stdout);
#else
	printf("       pinned memory : no \n"); fflush(stdout);
#endif

#if defined(__MAGMA)
	printf("       MAGMA         : yes \n"); fflush(stdout);
#else
	printf("       MAGMA         : no \n"); fflush(stdout);
#endif

#if defined(__PARA) && defined(__USE_3D_FFT)
	printf("       USE_3D_FFT    : yes (check size(pool)=1) \n"); fflush(stdout);
#else
	printf("       USE_3D_FFT    : no \n"); fflush(stdout);
#endif

#if defined(__DISABLE_CUDA_ADDUSDENS) || defined(__DISABLE_CUDA_VLOCPSI) || defined(__DISABLE_CUDA_NEWD) || defined(__PHIGEMM_DISABLE_SPECIALK)
	printf("       # DEBUG MODE #\n");fflush(stdout);
#if defined(__DISABLE_CUDA_ADDUSDENS)
	printf("         CUDA addusdens    = disabled\n");fflush(stdout);
#else
	printf("         CUDA addusdens    = enabled\n");fflush(stdout);
#endif
#if defined(__DISABLE_CUDA_VLOCPSI)
	printf("         CUDA vloc_psi     = disabled\n");fflush(stdout);
#else
	printf("         CUDA vloc_psi     = enabled\n");fflush(stdout);
#endif
#if defined(__DISABLE_CUDA_NEWD)
	printf("         CUDA newd         = disabled\n");fflush(stdout);
#else
	printf("         CUDA newd         = enabled\n");fflush(stdout);
#endif
#if defined(__PHIGEMM_DISABLE_SPECIALK)
	printf("         phiGEMM special-k = disabled\n");fflush(stdout);
#else
	printf("         phiGEMM special-k = enabled\n");fflush(stdout);
#endif
#endif

#endif

    printf("\n"); fflush(stdout);
	printf("     *******************************************************************\n"); fflush(stdout);
	printf("\n"); fflush(stdout);

#if defined(__PARA)
	}
#endif
}

extern "C"  void initcudaenv_()
{

	// In case of serial (default)
	int lRankThisNode = 0, lSizeThisNode = 1, lRank = -1;

#if defined(__PARA)
	paralleldetect_(&lRankThisNode, &lSizeThisNode, &lRank);
#endif

	gpubinding_(lRankThisNode, lSizeThisNode, lRank);

	preallocatedevicememory_(lRank);

#if defined(__PHIGEMM)
	initphigemm_(lRank);
#endif
}

void deallocatedevicememory_(){

	int ierr = 0;

#if defined(__CUDA_DEBUG)
	int i;
	size_t free, total;
#endif

	ierr = hipFree ( dev_scratch_QE[0] );

	if(ierr != hipSuccess) {
		fprintf( stderr, "\nError in memory release, program will be terminated!!! Bye...\n\n" );
		exit(EXIT_FAILURE);
	}

#if defined(__CUDA_DEBUG)
		hipMemGetInfo((size_t*)&free,(size_t*)&total);
		for (i = 0; i < ngpus_per_process; i++) {
			printf("[GPU %d] free: %lu (total: %lu)\n", qe_gpu_bonded[i], (unsigned long)free, (unsigned long)total); fflush(stdout);
		}
#endif

}

extern "C" void closecudaenv_()
{
#if !defined(__PHIGEMM_NOALLOC)
	deallocatedevicememory_();
#endif

#if defined(__PHIGEMM)
	phiGemmShutdown();
#endif

}
