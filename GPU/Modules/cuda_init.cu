#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2001-2013 Quantum ESPRESSO Foundation
 *
 * This file is distributed under the terms of the
 * GNU General Public License. See the file `License'
 * in the root directory of the present distribution,
 * or http://www.gnu.org/copyleft/gpl.txt .
 *
 */

#include <stdlib.h>
#include <stdio.h>

#include <hip/driver_types.h>

#if defined(__TIMELOG)
#include <time.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#endif

#include "cuda_env.h"

#if defined(__PHIGEMM)
#include "phigemm.h"
#endif

qeCudaMemDevPtr qe_dev_scratch;
qeCudaMemDevPtr qe_dev_zero_scratch;

qeCudaMemSizes qe_gpu_mem_tot;
qeCudaMemSizes qe_gpu_mem_unused;

qeCudaDevicesBond qe_gpu_bonded;

hipStream_t  qecudaStreams[ MAX_QE_GPUS ];
hipblasHandle_t qecudaHandles[ MAX_QE_GPUS ];

// Pre-loaded data-structure
int * preloaded_nlsm_D, * preloaded_nls_D;

// FFT plans (works only with "-D__CUDA_NOALLOC")
hipfftHandle qeCudaFFT_dfftp, qeCudaFFT_dffts;

// global useful information
long ngpus_detected;
long ngpus_used;
long ngpus_per_process;
long procs_per_gpu;


#if defined(__TIMELOG)
double cuda_cclock(void)
{
	struct timeval tv;
	struct timezone tz;
	double t;

	gettimeofday(&tv, &tz);

	t = (double)tv.tv_sec;
	t += ((double)tv.tv_usec)/1000000.0;

	return t;
}
#endif


void gpubinding_(int lRankThisNode, int lSizeThisNode, int lRank){

	int lNumDevicesThisNode = 0;
	int i;

#if defined(__PARA)

	/* Attach all MPI processes on this node to the available GPUs
	 * in round-robin fashion
	 */
	hipGetDeviceCount(&lNumDevicesThisNode);

	if (lNumDevicesThisNode == 0 && lRankThisNode == 0)
	{
		printf("***ERROR: no CUDA-capable devices were found.\n");
//		MPI_Abort( MPI_COMM_WORLD, EXIT_FAILURE );
		exit(EXIT_FAILURE);
	}

	ngpus_detected = lNumDevicesThisNode;

	if ( (lSizeThisNode % lNumDevicesThisNode ) != 0  )
	{
		printf("***WARNING: unbalanced configuration (%d MPI per node, %d GPUs per node)\n", lSizeThisNode, lNumDevicesThisNode);
		fflush(stdout);
	}

	if (ngpus_detected <= lSizeThisNode ){
		/* if GPUs are less then (or equal of) the number of  MPI processes on a single node,
		 * then PWscf uses all the GPU and one single GPU is assigned to one or multiple MPI processes with overlapping. */
		ngpus_used = ngpus_detected;
		ngpus_per_process = 1;
	} else {
		/* multi-GPU in parallel calculations is allowed ONLY if CUDA >= 4.0 */

		/* if GPUs are more than the MPI processes on a single node,
		 * then PWscf uses all the GPU and one or more GPUs are assigned
		 * to every single MPI processes without overlapping.
		 * *** NOT IMPLEMENTED YET ***
		 */
		ngpus_used = ngpus_detected;
		ngpus_per_process = 1;
	}

	procs_per_gpu = (lSizeThisNode < lNumDevicesThisNode) ? lSizeThisNode : lSizeThisNode / lNumDevicesThisNode;

	for (i = 0; i < ngpus_per_process; i++) {

		qe_gpu_bonded[i] = lRankThisNode % lNumDevicesThisNode;

#if defined(__CUDA_DEBUG)
		printf("Binding GPU %d on node of rank: %d (internal rank:%d)\n", qe_gpu_bonded[i], lRank, lRankThisNode); fflush(stdout);
#endif

	}

#else

	procs_per_gpu = 1;

	hipGetDeviceCount(&lNumDevicesThisNode);

	if (lNumDevicesThisNode == 0)
	{
		fprintf( stderr,"***ERROR*** no CUDA-capable devices were found on the machine.\n");
		exit(EXIT_FAILURE);
	}

	ngpus_detected = lNumDevicesThisNode;

	/* multi-GPU in serial calculations is allowed ONLY if CUDA >= 4.0 */
#if defined(__MULTI_GPU)
	ngpus_used = ngpus_per_process = lNumDevicesThisNode;
#else
	ngpus_used = ngpus_per_process = 1;
#endif

	for (i = 0; i < ngpus_per_process; i++) {
		/* NOTE: qe_gpu_bonded[0] is ALWAYS the main device for non multi-GPU
		 *       kernels.
		 */
		qe_gpu_bonded[i] = i;
	}

#endif
}

#if defined(__PHIGEMM)
extern "C" void initphigemm_(int lRank){
	/* Compatibility with CUDA 4.x (latest phiGEMM): */

#if defined(__CUDA_NOALLOC)
//	phiGemmInit(ngpus_per_process , NULL, (qeCudaMemSizes*)&qe_gpu_mem_tot, (int *)qe_gpu_bonded, lRank);
	phiGemmInit(ngpus_per_process , NULL, NULL, (int *)qe_gpu_bonded, lRank);
#else
	phiGemmInit(ngpus_per_process , (qeCudaMemDevPtr*)&qe_dev_scratch, (qeCudaMemSizes*)&qe_gpu_mem_tot, (int *)qe_gpu_bonded, lRank);
#endif
}
#endif

extern "C" void preallocatedevicememory_(int lRank){

	int ierr = 0;
	int i;

	size_t free, total;

	preloaded_nls_D = NULL;
	preloaded_nlsm_D = NULL;

	for (i = 0; i < ngpus_per_process; i++) {

		/* query the real free memory, taking into account the "stack" */
		if ( hipSetDevice(qe_gpu_bonded[i]) != hipSuccess) {
			printf("*** ERROR *** hipSetDevice(%d) failed!", qe_gpu_bonded[i] ); fflush(stdout);
			exit(EXIT_FAILURE);
		}

		qe_gpu_mem_tot[i] = (size_t) 0;

		ierr = hipMalloc ( (void**) &(qe_dev_scratch[i]), qe_gpu_mem_tot[i] );
		if ( ierr != hipSuccess) {
			fprintf( stderr, "\nError in (first zero) memory allocation , program will be terminated!!! Bye...\n\n");
			exit(EXIT_FAILURE);
		}

#if defined(__PARA)
	}

	mybarrier_();

	for (i = 0; i < ngpus_per_process; i++) {

		if ( hipSetDevice(qe_gpu_bonded[i]) != hipSuccess) {
			printf("*** ERROR *** hipSetDevice(%d) failed!", qe_gpu_bonded[i] ); fflush(stdout);
			exit(EXIT_FAILURE);
		}
#endif

		hipMemGetInfo((size_t*)&free,(size_t*)&total);

#if defined(__CUDA_DEBUG)
#if defined(__PARA)
		printf("[GPU %d - rank: %d] before: %lu (total: %lu)\n", qe_gpu_bonded[i], lRank, (unsigned long)free, (unsigned long)total); fflush(stdout);
#else
		printf("[GPU %d] before: %lu (total: %lu)\n", qe_gpu_bonded[i], (unsigned long)free, (unsigned long)total); fflush(stdout);
#endif
#endif

		qe_gpu_mem_tot[i] = (size_t) ((free * __SCALING_MEM_FACTOR__ ) / procs_per_gpu);
		qe_gpu_mem_unused[i] = qe_gpu_mem_tot[i];


#if !defined(__CUDA_NOALLOC)
		/* Do real allocation */
		ierr = hipMalloc ( (void**) &(qe_dev_scratch[i]), (size_t) qe_gpu_mem_tot[i] );
		if ( ierr != hipSuccess) {
			fprintf( stderr, "\nError in memory allocation, program will be terminated (%d)!!! Bye...\n\n", ierr );
			exit(EXIT_FAILURE);
		}

		qe_dev_zero_scratch[i] = qe_dev_scratch[i];
#endif

#if defined(__PARA) && defined(__CUDA_DEBUG)
	}

	mybarrier_();

	for (i = 0; i < ngpus_per_process; i++) {
#endif

#if defined(__CUDA_DEBUG)
#if defined(__PARA)
		printf("[GPU %d - rank: %d] after: %lu (total: %lu)\n", qe_gpu_bonded[i], lRank, (unsigned long)free, (unsigned long)total);
		fflush(stdout);
#else
		printf("[GPU %d] after: %lu (total: %lu)\n", qe_gpu_bonded[i], (unsigned long)free, (unsigned long)total);
		fflush(stdout);
#endif
#endif
	}
	
#if defined(__PARA)
	// Is this necessary?
	mybarrier_();
#endif

	// Print CUDA header
	print_cuda_header_(lRank);
}

extern "C"  void initStreams_()
{
	int ierr, i;

	for (i = 0; i < ngpus_per_process; i++) {
		ierr = hipStreamCreate( &qecudaStreams[ i ] );
		qecudaGenericErr((hipError_t) ierr, "INIT_CUDA", "error during stream creation");

		if ( hipblasCreate( &qecudaHandles[ i ] ) != HIPBLAS_STATUS_SUCCESS ) {
			printf("\n*** CUDA VLOC_PSI_K *** ERROR *** cublasInit() for device %d failed!",qe_gpu_bonded[i]);
			fflush(stdout);
			exit(EXIT_FAILURE);
		}
	}
}

extern "C"  void initcudaenv_()
{
	// In case of serial (default)
	int lRankThisNode = 0, lSizeThisNode = 1, lRank = -1;

#if defined(__PARA)
	paralleldetect_(&lRankThisNode, &lSizeThisNode, &lRank);
#endif

	gpubinding_(lRankThisNode, lSizeThisNode, lRank);

	preallocatedevicememory_(lRank);

#if defined(__PHIGEMM)
	initphigemm_(lRank);
#endif

	initStreams_();
}

void deallocatedevicememory_(){

	int ierr = 0;

	// Assumed 1 GPU per process...
	ierr = hipFree ( qe_dev_scratch[0] );

	if(ierr != hipSuccess) {
		fprintf( stderr, "\nError in memory release, program will be terminated!!! Bye...\n\n" );
		exit(EXIT_FAILURE);
	}

	return;
}

extern "C"  void destroyStreams_()
{
	int ierr, i;

	for (i = 0; i < ngpus_per_process; i++) {
		ierr = hipStreamDestroy( qecudaStreams[ i ] );
		qecudaGenericErr((hipError_t) ierr, "INIT_CUDA", "error during stream creation");

		if ( hipblasDestroy( qecudaHandles[ i ] ) != HIPBLAS_STATUS_SUCCESS ) {
			printf("\n*** CUDA INIT_CUDA *** ERROR *** hipblasDestroy() for device %d failed!",qe_gpu_bonded[i]);
			fflush(stdout);
			exit(EXIT_FAILURE);
		}
	}
}

extern "C" void closecudaenv_()
{
#if !defined(__CUDA_NOALLOC)
	deallocatedevicememory_();
#endif

	destroyStreams_();

#if defined(__PHIGEMM)
	phiGemmShutdown();
#endif

}
